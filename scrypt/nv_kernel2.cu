#include "hip/hip_runtime.h"
//
// Experimental Kernel for Kepler (Compute 3.5) devices
// code submitted by nVidia performance engineer Alexey Panteleev
// with modifications by Christian Buchner
//
// for Compute 3.5
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=80
// for Compute 3.0
// NOTE: compile this .cu module for compute_30,sm_30 with --maxrregcount=63
//

#include <map>

#include "hip/hip_runtime.h"
#include "miner.h"

#include "salsa_kernel.h"
#include "nv_kernel2.h"
#include "cuda_helper.h"

#define THREADS_PER_WU 1  // single thread per hash

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define __ldg(x) (*(x))
#endif

// grab lane ID
static __device__ __inline__ uint32_t __laneId() { uint32_t laneId; asm( "mov.u32 %0, %%laneid;" : "=r"( laneId ) ); return laneId; }

// forward references
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, uint32_t LOOKUP_GAP);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, uint32_t LOOKUP_GAP);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1; // N - 1
__constant__ uint32_t c_spacing; // (N+LOOKUP_GAP-1)/LOOKUP_GAP


NV2Kernel::NV2Kernel() : KernelInterface()
{
}

void NV2Kernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool NV2Kernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, uint32_t N, uint32_t LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;

	// make some constants available to kernel, update only initially and when changing
	static int prev_N[MAX_DEVICES] = {0};
	if (N != prev_N[thr_id]) {
		uint32_t h_N = N;
		uint32_t h_N_1 = N-1;
		uint32_t h_spacing = (N+LOOKUP_GAP-1)/LOOKUP_GAP;

		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_spacing), &h_spacing, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);

		prev_N[thr_id] = N;
	}

	// First phase: Sequential writes to scratchpad.
	const int batch = device_batchsize[thr_id];
	uint32_t pos = 0;

	do
	{
		if (LOOKUP_GAP == 1) {
			if (IS_SCRYPT())      nv2_scrypt_core_kernelA<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
			if (IS_SCRYPT_JANE()) nv2_scrypt_core_kernelA<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
		} else {
			if (IS_SCRYPT())      nv2_scrypt_core_kernelA_LG<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (IS_SCRYPT_JANE()) nv2_scrypt_core_kernelA_LG<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
		}
		pos += batch;
	} while (pos < N);

	// Second phase: Random read access from scratchpad.
	pos = 0;
	do
	{
		if (LOOKUP_GAP == 1) {
			if (IS_SCRYPT())      nv2_scrypt_core_kernelB<A_SCRYPT     > <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			if (IS_SCRYPT_JANE()) nv2_scrypt_core_kernelB<A_SCRYPT_JANE> <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
		} else {
			if (IS_SCRYPT())      nv2_scrypt_core_kernelB_LG<A_SCRYPT     > <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (IS_SCRYPT_JANE()) nv2_scrypt_core_kernelB_LG<A_SCRYPT_JANE> <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
		}

		pos += batch;
	} while (pos < N);

	return success;
}

//static __device__ uint4& operator^=(uint4& left, const uint4& right)
//{
//	left.x ^= right.x;
//	left.y ^= right.y;
//	left.z ^= right.z;
//	left.w ^= right.w;
//	return left;
//}

__device__ __forceinline__ uint4 __shfl(const uint4 val, uint32_t lane, uint32_t width)
{
	return make_uint4(
		(uint32_t)__shfl((int)val.x, lane, width),
		(uint32_t)__shfl((int)val.y, lane, width),
		(uint32_t)__shfl((int)val.z, lane, width),
		(uint32_t)__shfl((int)val.w, lane, width));
}

__device__ __forceinline__ void __transposed_write_BC(uint4 (&B)[4], uint4 (&C)[4], uint4 *D, int spacing)
{
	uint32_t laneId = __laneId();

	uint32_t lane8 = laneId&7;
	uint32_t tile  = laneId/8;

	uint4 T1[8], T2[8];

	/* Source matrix, A-H are threads, 0-7 are data items, thread A is marked with `*`:

	   *A0  B0  C0  D0  E0  F0  G0  H0
	   *A1  B1  C1  D1  E1  F1  G1  H1
	   *A2  B2  C2  D2  E2  F2  G2  H2
	   *A3  B3  C3  D3  E3  F3  G3  H3
	   *A4  B4  C4  D4  E4  F4  G4  H4
	   *A5  B5  C5  D5  E5  F5  G5  H5
	   *A6  B6  C6  D6  E6  F6  G6  H6
	   *A7  B7  C7  D7  E7  F7  G7  H7
	*/

	// rotate rows
	T1[0] = B[0];
	T1[1] = __shfl(B[1], lane8 + 7, 8);
	T1[2] = __shfl(B[2], lane8 + 6, 8);
	T1[3] = __shfl(B[3], lane8 + 5, 8);
	T1[4] = __shfl(C[0], lane8 + 4, 8);
	T1[5] = __shfl(C[1], lane8 + 3, 8);
	T1[6] = __shfl(C[2], lane8 + 2, 8);
	T1[7] = __shfl(C[3], lane8 + 1, 8);

	/* Matrix after row rotates:

	   *A0  B0  C0  D0  E0  F0  G0  H0
		H1 *A1  B1  C1  D1  E1  F1  G1
		G2  H2 *A2  B2  C2  D2  E2  F2
		F3  G3  H3 *A3  B3  C3  D3  E3
		E4  F4  G4  H4 *A4  B4  C4  D4
		D5  E5  F5  G5  H5 *A5  B5  C5
		C6  D6  E6  F6  G6  H6 *A6  B6
		B7  C7  D7  E7  F7  G7  H7 *A7
	*/

	// rotate columns up using a barrel shifter simulation
	// column X is rotated up by (X+1) items
#pragma unroll 8
	for (int n = 0; n < 8; n++) T2[n] = ((lane8 + 1) & 1) ? T1[(n + 1) &7] : T1[n];
#pragma unroll 8
	for (int n = 0; n < 8; n++) T1[n] = ((lane8 + 1) & 2) ? T2[(n + 2) &7] : T2[n];
#pragma unroll 8
	for (int n = 0; n < 8; n++) T2[n] = ((lane8 + 1) & 4) ? T1[(n + 4) & 7] : T1[n];

	/* Matrix after column rotates:

		H1  H2  H3  H4  H5  H6  H7  H0
		G2  G3  G4  G5  G6  G7  G0  G1
		F3  F4  F5  F6  F7  F0  F1  F2
		E4  E5  E6  E7  E0  E1  E2  E3
		D5  D6  D7  D0  D1  D2  D3  D4
		C6  C7  C0  C1  C2  C3  C4  C5
		B7  B0  B1  B2  B3  B4  B5  B6
	   *A0 *A1 *A2 *A3 *A4 *A5 *A6 *A7
	*/

	// rotate rows again using address math and write to D, in reverse row order
	D[spacing*2*(32*tile   )+ lane8     ] = T2[7];
	D[spacing * 2 * (32 * tile + 4) + (lane8 + 7) & 7] = T2[6];
	D[spacing * 2 * (32 * tile + 8) + (lane8 + 6) & 7] = T2[5];
	D[spacing * 2 * (32 * tile + 12) + (lane8 + 5) & 7] = T2[4];
	D[spacing * 2 * (32 * tile + 16) + (lane8 + 4) & 7] = T2[3];
	D[spacing * 2 * (32 * tile + 20) + (lane8 + 3) & 7] = T2[2];
	D[spacing * 2 * (32 * tile + 24) + (lane8 + 2) & 7] = T2[1];
	D[spacing * 2 * (32 * tile + 28) + (lane8 + 1) & 7] = T2[0];
}

__device__ __forceinline__ void __transposed_read_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	uint32_t laneId = __laneId();

	uint32_t lane8 = laneId & 7;
	uint32_t tile  = laneId/8;

	// Perform the same transposition as in __transposed_write_BC, but in reverse order.
	// See the illustrations in comments for __transposed_write_BC.

	// read and rotate rows, in reverse row order
	uint4 T1[8], T2[8];
	T1[7] = __ldg(&S[(spacing*2*(32*tile   ) +  lane8      + 8*__shfl(row, 0, 8))]);
	T1[6] = __ldg(&S[(spacing * 2 * (32 * tile + 4) + (lane8 + 7) & 7 + 8 * __shfl(row, 1, 8))]);
	T1[5] = __ldg(&S[(spacing * 2 * (32 * tile + 8) + (lane8 + 6) & 7 + 8 * __shfl(row, 2, 8))]);
	T1[4] = __ldg(&S[(spacing * 2 * (32 * tile + 12) + (lane8 + 5) & 7 + 8 * __shfl(row, 3, 8))]);
	T1[3] = __ldg(&S[(spacing * 2 * (32 * tile + 16) + (lane8 + 4) & 7 + 8 * __shfl(row, 4, 8))]);
	T1[2] = __ldg(&S[(spacing * 2 * (32 * tile + 20) + (lane8 + 3) & 7 + 8 * __shfl(row, 5, 8))]);
	T1[1] = __ldg(&S[(spacing * 2 * (32 * tile + 24) + (lane8 + 2) & 7 + 8 * __shfl(row, 6, 8))]);
	T1[0] = __ldg(&S[(spacing * 2 * (32 * tile + 28) + (lane8 + 1) & 7 + 8 * __shfl(row, 7, 8))]);

	// rotate columns down using a barrel shifter simulation
	// column X is rotated down by (X+1) items, or up by (8-(X+1)) = (7-X) items
#pragma unroll 8
	for (int n = 0; n < 8; n++) T2[n] = ((7 - lane8) & 1) ? T1[(n + 1) & 7] : T1[n];
#pragma unroll 8
	for (int n = 0; n < 8; n++) T1[n] = ((7 - lane8) & 2) ? T2[(n + 2) &7] : T2[n];
#pragma unroll 8
	for (int n = 0; n < 8; n++) T2[n] = ((7 - lane8) & 4) ? T1[(n + 4) & 7] : T1[n];

	// rotate rows
	B[0] = T2[0];
	B[1] = __shfl(T2[1], lane8 + 1, 8);
	B[2] = __shfl(T2[2], lane8 + 2, 8);
	B[3] = __shfl(T2[3], lane8 + 3, 8);
	C[0] = __shfl(T2[4], lane8 + 4, 8);
	C[1] = __shfl(T2[5], lane8 + 5, 8);
	C[2] = __shfl(T2[6], lane8 + 6, 8);
	C[3] = __shfl(T2[7], lane8 + 7, 8);

}

__device__ __forceinline__ void __transposed_xor_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	uint4 BT[4], CT[4];
	__transposed_read_BC(S, BT, CT, spacing, row);

#pragma unroll 4
	for(int n = 0; n < 4; n++)
	{
		B[n] ^= BT[n];
		C[n] ^= CT[n];
	}
}

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define ROTL(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
	// Kepler (Compute 3.5)
	#define ROTL(a, b) __funnelshift_l( a, a, b );
#endif



#if 0

#define QUARTER(a,b,c,d) \
	a += b; d ^= a; d = ROTL(d,16); \
	c += d; b ^= c; b = ROTL(b,12); \
	a += b; d ^= a; d = ROTL(d,8); \
	c += d; b ^= c; b = ROTL(b,7);

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#else

#define ADD4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 += s1; d2 += s2; d3 += s3; d4 += s4;

#define XOR4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 ^= s1; d2 ^= s2; d3 ^= s3; d4 ^= s4;

#define ROTL4(d1,d2,d3,d4,amt) \
	d1 = ROTL(d1, amt); d2 = ROTL(d2, amt); d3 = ROTL(d3, amt); d4 = ROTL(d4, amt);

#define QROUND(a1,a2,a3,a4, b1,b2,b3,b4, c1,c2,c3,c4, amt) \
	ADD4 (a1,a2,a3,a4, c1,c2,c3,c4) \
	XOR4 (b1,b2,b3,b4, a1,a2,a3,a4) \
	ROTL4(b1,b2,b3,b4, amt)

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#endif


#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 7); a1^=ROTL(a10, 7); a2^=ROTL(a20, 7); a3^=ROTL(a30, 7);\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 9); a1^=ROTL(a10, 9); a2^=ROTL(a20, 9); a3^=ROTL(a30, 9);\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 13); a1^=ROTL(a10, 13); a2^=ROTL(a20, 13); a3^=ROTL(a30, 13);\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 18); a1^=ROTL(a10, 18); a2^=ROTL(a20, 18); a3^=ROTL(a30, 18);\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}


template <int ALGO> static __device__ void block_mixer(uint4 *B, uint4 *C)
{
  switch (ALGO)
  {
	case A_SCRYPT:      xor_salsa8(B, C); break;
	case A_SCRYPT_JANE: xor_chacha8(B, C); break;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Experimental Scrypt core kernel for Titan devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_N);
		++i;
	} else
		__transposed_read_BC((uint4*)(V + (i-1)*32), B, C, c_N, 0);

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		__transposed_write_BC(B, C, (uint4*)(V + i*32), c_N);
		++i;
	}
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, uint32_t LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_spacing);
		++i;
	} else {
		int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
		__transposed_read_BC((uint4*)(V + pos*32), B, C, c_spacing, 0);
		while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	}

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		if (i % LOOKUP_GAP == 0)
		  __transposed_write_BC(B, C, (uint4*)(V + (i/LOOKUP_GAP)*32), c_spacing);
		++i;
	}
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
		__transposed_read_BC((uint4*)V, B, C, c_N, c_N_1);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	} else
		__transposed_read_BC((uint4*)g_odata, B, C, 1, 0);

	for (int i = begin; i < end; i++)  {
		int slot = C[0].x & c_N_1;
		__transposed_xor_BC((uint4*)(V), B, C, c_N, slot);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, uint32_t LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
	  int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
	  __transposed_read_BC((uint4*)V, B, C, c_spacing, pos);
	  while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	} else {
		__transposed_read_BC((uint4*)g_odata, B, C, 1, 0);
	}

	for (int i = begin; i < end; i++)  
	{
		int slot = C[0].x & c_N_1;
		int pos = slot/LOOKUP_GAP, loop = slot-pos*LOOKUP_GAP;
		uint4 b[4], c[4]; __transposed_read_BC((uint4*)(V), b, c, c_spacing, pos);
		while(loop--) { block_mixer<ALGO>(b, c); block_mixer<ALGO>(c, b); }
#pragma unroll 4
		for(int n = 0; n < 4; n++) { B[n] ^= b[n]; C[n] ^= c[n]; }
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}


//
// Maxcoin related Keccak implementation (Keccak256)
//

// from salsa_kernel.cu
extern std::map<int, int> context_blocks;
extern std::map<int, int> context_wpb;
extern std::map<int, KernelInterface *> context_kernel;
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

__constant__ uint64_t ptarget64[4];

// ROL macro replaced with the inline assembly code below to work around a performance issue
//#define ROL(a, offset) ((((uint64_t)a) << ((offset) % 64)) ^ (((uint64_t)a) >> (64-((offset) % 64))))
__inline__ __device__ uint2 ROL(const uint2 a, const int offset) {
	uint2 result;
	if(offset >= 32) {
		asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
		asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
	} else {
		asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
	return result;
}
#define ROL_mult8(a, offset) ROL(a, offset)

//__inline__ __device__ uint64_t devectorize(uint2 v) { return __double_as_longlong(__hiloint2double(v.y, v.x)); }
//__inline__ __device__ uint2 vectorize(uint64_t v) { return make_uint2(__double2loint(__longlong_as_double(v)), __double2hiint(__longlong_as_double(v))); }
//__inline__ __device__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
//__inline__ __device__ uint2 operator& (uint2 a, uint2 b) { return make_uint2(a.x & b.x, a.y & b.y); }
//__inline__ __device__ uint2 operator| (uint2 a, uint2 b) { return make_uint2(a.x | b.x, a.y | b.y); }
//__inline__ __device__ uint2 operator~ (uint2 a) { return make_uint2(~a.x, ~a.y); }
//__inline__ __device__ void operator^= (uint2 &a, uint2 b) { a = a ^ b; }

__constant__ uint64_t KeccakF_RoundConstants[24];

static uint64_t host_KeccakF_RoundConstants[24] =
{
	(uint64_t)0x0000000000000001ULL,
	(uint64_t)0x0000000000008082ULL,
	(uint64_t)0x800000000000808aULL,
	(uint64_t)0x8000000080008000ULL,
	(uint64_t)0x000000000000808bULL,
	(uint64_t)0x0000000080000001ULL,
	(uint64_t)0x8000000080008081ULL,
	(uint64_t)0x8000000000008009ULL,
	(uint64_t)0x000000000000008aULL,
	(uint64_t)0x0000000000000088ULL,
	(uint64_t)0x0000000080008009ULL,
	(uint64_t)0x000000008000000aULL,
	(uint64_t)0x000000008000808bULL,
	(uint64_t)0x800000000000008bULL,
	(uint64_t)0x8000000000008089ULL,
	(uint64_t)0x8000000000008003ULL,
	(uint64_t)0x8000000000008002ULL,
	(uint64_t)0x8000000000000080ULL,
	(uint64_t)0x000000000000800aULL,
	(uint64_t)0x800000008000000aULL,
	(uint64_t)0x8000000080008081ULL,
	(uint64_t)0x8000000000008080ULL,
	(uint64_t)0x0000000080000001ULL,
	(uint64_t)0x8000000080008008ULL
};

__constant__ uint64_t pdata64[10];

//static __device__ __forceinline__ uint32_t cuda_swab32(uint32_t x)
//{
//	return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
//		  | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
//}

// in this implementation the first and last iteration of the for() loop were explicitly
// unrolled and redundant operations were removed (e.g. operations on zero inputs, and
// computation of unnecessary outputs)
__global__ __launch_bounds__(256, 2)
void titan_crypto_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
	uint2 Aba, Abe, Abi, Abo, Abu;
	uint2 Aga, Age, Agi, Ago, Agu;
	uint2 Aka, Ake, Aki, Ako, Aku;
	uint2 Ama, Ame, Ami, Amo, Amu;
	uint2 Asa, Ase, Asi, Aso, Asu;
	uint2 BCa, BCe, BCi, BCo, BCu;
	uint2 Da, De, Di, Do, Du;
	uint2 Eba, Ebe, Ebi, Ebo, Ebu;
	uint2 Ega, Ege, Egi, Ego, Egu;
	uint2 Eka, Eke, Eki, Eko, Eku;
	uint2 Ema, Eme, Emi, Emo, Emu;
	uint2 Esa, Ese, Esi, Eso, Esu;

	// embed unique nonce into source data stream in pdata[]
	Agu = vectorize((pdata64[9] & 0x00000000FFFFFFFFULL) | (((uint64_t)cuda_swab32(nonce + ((blockIdx.x * blockDim.x) + threadIdx.x))) << 32));

	//    prepareTheta
	BCa = vectorize(pdata64[0]^pdata64[5]^0x0000000000000001ULL);
	BCe = vectorize(pdata64[1]^pdata64[6]^0x8000000000000000ULL);
	BCi = vectorize(pdata64[2]^pdata64[7]);
	BCo = vectorize(pdata64[3]^pdata64[8]);
	BCu = vectorize(pdata64[4])^Agu;

	//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
	Da = BCu^ROL(BCe, 1);
	De = BCa^ROL(BCi, 1);
	Di = BCe^ROL(BCo, 1);
	Do = BCi^ROL(BCu, 1);
	Du = BCo^ROL(BCa, 1);

	Aba = vectorize(pdata64[0]) ^ Da;
	BCa = Aba;
	Age = vectorize(pdata64[6]) ^ De;
	BCe = ROL(Age, 44);
	Aki = Di;
	BCi = ROL(Aki, 43);
	Amo = Do;
	BCo = ROL(Amo, 21);
	Asu = Du;
	BCu = ROL(Asu, 14);
	Eba =   BCa ^((~BCe)&  BCi );
	Eba ^= vectorize((uint64_t)KeccakF_RoundConstants[0]);
	Ebe =   BCe ^((~BCi)&  BCo );
	Ebi =   BCi ^((~BCo)&  BCu );
	Ebo =   BCo ^((~BCu)&  BCa );
	Ebu =   BCu ^((~BCa)&  BCe );

	Abo = vectorize(pdata64[3]) ^ Do;
	BCa = ROL(Abo, 28);
	Agu ^= Du;
	BCe = ROL(Agu, 20);
	Aka = vectorize(0x0000000000000001ULL) ^ Da;
	BCi = ROL(Aka,  3);
	Ame = vectorize(0x8000000000000000ULL) ^ De;
	BCo = ROL(Ame, 45);
	Asi = Di;
	BCu = ROL(Asi, 61);
	Ega =   BCa ^((~BCe)&  BCi );
	Ege =   BCe ^((~BCi)&  BCo );
	Egi =   BCi ^((~BCo)&  BCu );
	Ego =   BCo ^((~BCu)&  BCa );
	Egu =   BCu ^((~BCa)&  BCe );

	Abe = vectorize(pdata64[1]) ^ De;
	BCa = ROL(Abe,  1);
	Agi = vectorize(pdata64[7]) ^ Di;
	BCe = ROL(Agi,  6);
	Ako = Do;
	BCi = ROL(Ako, 25);
	Amu = Du;
	BCo = ROL(Amu,  8);
	Asa = Da;
	BCu = ROL(Asa, 18);
	Eka =   BCa ^((~BCe)&  BCi );
	Eke =   BCe ^((~BCi)&  BCo );
	Eki =   BCi ^((~BCo)&  BCu );
	Eko =   BCo ^((~BCu)&  BCa );
	Eku =   BCu ^((~BCa)&  BCe );

	Abu = vectorize(pdata64[4]) ^ Du;
	BCa = ROL(Abu, 27);
	Aga = vectorize(pdata64[5]) ^ Da;
	BCe = ROL(Aga, 36);
	Ake = De;
	BCi = ROL(Ake, 10);
	Ami = Di;
	BCo = ROL(Ami, 15);
	Aso = Do;
	BCu = ROR8(Aso);
	Ema =   BCa ^((~BCe)&  BCi );
	Eme =   BCe ^((~BCi)&  BCo );
	Emi =   BCi ^((~BCo)&  BCu );
	Emo =   BCo ^((~BCu)&  BCa );
	Emu =   BCu ^((~BCa)&  BCe );

	Abi = vectorize(pdata64[2]) ^ Di;
	BCa = ROL(Abi, 62);
	Ago = vectorize(pdata64[8]) ^ Do;
	BCe = ROL(Ago, 55);
	Aku = Du;
	BCi = ROL(Aku, 39);
	Ama = Da;
	BCo = ROL(Ama, 41);
	Ase = De;
	BCu = ROL(Ase,  2);
	Esa =   BCa ^((~BCe)&  BCi );
	Ese =   BCe ^((~BCi)&  BCo );
	Esi =   BCi ^((~BCo)&  BCu );
	Eso =   BCo ^((~BCu)&  BCa );
	Esu =   BCu ^((~BCa)&  BCe );

	//    prepareTheta
	BCa = Eba^Ega^Eka^Ema^Esa;
	BCe = Ebe^Ege^Eke^Eme^Ese;
	BCi = Ebi^Egi^Eki^Emi^Esi;
	BCo = Ebo^Ego^Eko^Emo^Eso;
	BCu = Ebu^Egu^Eku^Emu^Esu;

	//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
	Da = BCu^ROL(BCe, 1);
	De = BCa^ROL(BCi, 1);
	Di = BCe^ROL(BCo, 1);
	Do = BCi^ROL(BCu, 1);
	Du = BCo^ROL(BCa, 1);

	Eba ^= Da;
	BCa = Eba;
	Ege ^= De;
	BCe = ROL(Ege, 44);
	Eki ^= Di;
	BCi = ROL(Eki, 43);
	Emo ^= Do;
	BCo = ROL(Emo, 21);
	Esu ^= Du;
	BCu = ROL(Esu, 14);
	Aba =   BCa ^((~BCe)&  BCi );
	Aba ^= vectorize((uint64_t)KeccakF_RoundConstants[1]);
	Abe =   BCe ^((~BCi)&  BCo );
	Abi =   BCi ^((~BCo)&  BCu );
	Abo =   BCo ^((~BCu)&  BCa );
	Abu =   BCu ^((~BCa)&  BCe );

	Ebo ^= Do;
	BCa = ROL(Ebo, 28);
	Egu ^= Du;
	BCe = ROL(Egu, 20);
	Eka ^= Da;
	BCi = ROL(Eka, 3);
	Eme ^= De;
	BCo = ROL(Eme, 45);
	Esi ^= Di;
	BCu = ROL(Esi, 61);
	Aga =   BCa ^((~BCe)&  BCi );
	Age =   BCe ^((~BCi)&  BCo );
	Agi =   BCi ^((~BCo)&  BCu );
	Ago =   BCo ^((~BCu)&  BCa );
	Agu =   BCu ^((~BCa)&  BCe );

	Ebe ^= De;
	BCa = ROL(Ebe, 1);
	Egi ^= Di;
	BCe = ROL(Egi, 6);
	Eko ^= Do;
	BCi = ROL(Eko, 25);
	Emu ^= Du;
	BCo = ROL(Emu, 8);
	Esa ^= Da;
	BCu = ROL(Esa, 18);
	Aka =   BCa ^((~BCe)&  BCi );
	Ake =   BCe ^((~BCi)&  BCo );
	Aki =   BCi ^((~BCo)&  BCu );
	Ako =   BCo ^((~BCu)&  BCa );
	Aku =   BCu ^((~BCa)&  BCe );

	Ebu ^= Du;
	BCa = ROL(Ebu, 27);
	Ega ^= Da;
	BCe = ROL(Ega, 36);
	Eke ^= De;
	BCi = ROL(Eke, 10);
	Emi ^= Di;
	BCo = ROL(Emi, 15);
	Eso ^= Do;
	BCu = ROR8(Eso);
	Ama =   BCa ^((~BCe)&  BCi );
	Ame =   BCe ^((~BCi)&  BCo );
	Ami =   BCi ^((~BCo)&  BCu );
	Amo =   BCo ^((~BCu)&  BCa );
	Amu =   BCu ^((~BCa)&  BCe );

	Ebi ^= Di;
	BCa = ROL(Ebi, 62);
	Ego ^= Do;
	BCe = ROL(Ego, 55);
	Eku ^= Du;
	BCi = ROL(Eku, 39);
	Ema ^= Da;
	BCo = ROL(Ema, 41);
	Ese ^= De;
	BCu = ROL(Ese, 2);
	Asa =   BCa ^((~BCe)&  BCi );
	Ase =   BCe ^((~BCi)&  BCo );
	Asi =   BCi ^((~BCo)&  BCu );
	Aso =   BCo ^((~BCu)&  BCa );
	Asu =   BCu ^((~BCa)&  BCe );

	#pragma unroll 1
	for( int laneCount = 2; laneCount < 22; laneCount += 2 )
	{
		//    prepareTheta
		BCa = Aba^Aga^Aka^Ama^Asa;
		BCe = Abe^Age^Ake^Ame^Ase;
		BCi = Abi^Agi^Aki^Ami^Asi;
		BCo = Abo^Ago^Ako^Amo^Aso;
		BCu = Abu^Agu^Aku^Amu^Asu;

		//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
		Da = BCu^ROL(BCe, 1);
		De = BCa^ROL(BCi, 1);
		Di = BCe^ROL(BCo, 1);
		Do = BCi^ROL(BCu, 1);
		Du = BCo^ROL(BCa, 1);

		Aba ^= Da;
		BCa = Aba;
		Age ^= De;
		BCe = ROL(Age, 44);
		Aki ^= Di;
		BCi = ROL(Aki, 43);
		Amo ^= Do;
		BCo = ROL(Amo, 21);
		Asu ^= Du;
		BCu = ROL(Asu, 14);
		Eba =   BCa ^((~BCe)&  BCi );
		Eba ^= vectorize((uint64_t)KeccakF_RoundConstants[laneCount]);
		Ebe =   BCe ^((~BCi)&  BCo );
		Ebi =   BCi ^((~BCo)&  BCu );
		Ebo =   BCo ^((~BCu)&  BCa );
		Ebu =   BCu ^((~BCa)&  BCe );

		Abo ^= Do;
		BCa = ROL(Abo, 28);
		Agu ^= Du;
		BCe = ROL(Agu, 20);
		Aka ^= Da;
		BCi = ROL(Aka,  3);
		Ame ^= De;
		BCo = ROL(Ame, 45);
		Asi ^= Di;
		BCu = ROL(Asi, 61);
		Ega =   BCa ^((~BCe)&  BCi );
		Ege =   BCe ^((~BCi)&  BCo );
		Egi =   BCi ^((~BCo)&  BCu );
		Ego =   BCo ^((~BCu)&  BCa );
		Egu =   BCu ^((~BCa)&  BCe );

		Abe ^= De;
		BCa = ROL(Abe,  1);
		Agi ^= Di;
		BCe = ROL(Agi,  6);
		Ako ^= Do;
		BCi = ROL(Ako, 25);
		Amu ^= Du;
		BCo = ROL(Amu,  8);
		Asa ^= Da;
		BCu = ROL(Asa, 18);
		Eka =   BCa ^((~BCe)&  BCi );
		Eke =   BCe ^((~BCi)&  BCo );
		Eki =   BCi ^((~BCo)&  BCu );
		Eko =   BCo ^((~BCu)&  BCa );
		Eku =   BCu ^((~BCa)&  BCe );

		Abu ^= Du;
		BCa = ROL(Abu, 27);
		Aga ^= Da;
		BCe = ROL(Aga, 36);
		Ake ^= De;
		BCi = ROL(Ake, 10);
		Ami ^= Di;
		BCo = ROL(Ami, 15);
		Aso ^= Do;
		BCu = ROR8(Aso);
		Ema =   BCa ^((~BCe)&  BCi );
		Eme =   BCe ^((~BCi)&  BCo );
		Emi =   BCi ^((~BCo)&  BCu );
		Emo =   BCo ^((~BCu)&  BCa );
		Emu =   BCu ^((~BCa)&  BCe );

		Abi ^= Di;
		BCa = ROL(Abi, 62);
		Ago ^= Do;
		BCe = ROL(Ago, 55);
		Aku ^= Du;
		BCi = ROL(Aku, 39);
		Ama ^= Da;
		BCo = ROL(Ama, 41);
		Ase ^= De;
		BCu = ROL(Ase,  2);
		Esa =   BCa ^((~BCe)&  BCi );
		Ese =   BCe ^((~BCi)&  BCo );
		Esi =   BCi ^((~BCo)&  BCu );
		Eso =   BCo ^((~BCu)&  BCa );
		Esu =   BCu ^((~BCa)&  BCe );

		//    prepareTheta
		BCa = Eba^Ega^Eka^Ema^Esa;
		BCe = Ebe^Ege^Eke^Eme^Ese;
		BCi = Ebi^Egi^Eki^Emi^Esi;
		BCo = Ebo^Ego^Eko^Emo^Eso;
		BCu = Ebu^Egu^Eku^Emu^Esu;

		//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
		Da = BCu^ROL(BCe, 1);
		De = BCa^ROL(BCi, 1);
		Di = BCe^ROL(BCo, 1);
		Do = BCi^ROL(BCu, 1);
		Du = BCo^ROL(BCa, 1);

		Eba ^= Da;
		BCa = Eba;
		Ege ^= De;
		BCe = ROL(Ege, 44);
		Eki ^= Di;
		BCi = ROL(Eki, 43);
		Emo ^= Do;
		BCo = ROL(Emo, 21);
		Esu ^= Du;
		BCu = ROL(Esu, 14);
		Aba =   BCa ^((~BCe)&  BCi );
		Aba ^= vectorize((uint64_t)KeccakF_RoundConstants[laneCount+1]);
		Abe =   BCe ^((~BCi)&  BCo );
		Abi =   BCi ^((~BCo)&  BCu );
		Abo =   BCo ^((~BCu)&  BCa );
		Abu =   BCu ^((~BCa)&  BCe );

		Ebo ^= Do;
		BCa = ROL(Ebo, 28);
		Egu ^= Du;
		BCe = ROL(Egu, 20);
		Eka ^= Da;
		BCi = ROL(Eka, 3);
		Eme ^= De;
		BCo = ROL(Eme, 45);
		Esi ^= Di;
		BCu = ROL(Esi, 61);
		Aga =   BCa ^((~BCe)&  BCi );
		Age =   BCe ^((~BCi)&  BCo );
		Agi =   BCi ^((~BCo)&  BCu );
		Ago =   BCo ^((~BCu)&  BCa );
		Agu =   BCu ^((~BCa)&  BCe );

		Ebe ^= De;
		BCa = ROL(Ebe, 1);
		Egi ^= Di;
		BCe = ROL(Egi, 6);
		Eko ^= Do;
		BCi = ROL(Eko, 25);
		Emu ^= Du;
		BCo = ROL(Emu, 8);
		Esa ^= Da;
		BCu = ROL(Esa, 18);
		Aka =   BCa ^((~BCe)&  BCi );
		Ake =   BCe ^((~BCi)&  BCo );
		Aki =   BCi ^((~BCo)&  BCu );
		Ako =   BCo ^((~BCu)&  BCa );
		Aku =   BCu ^((~BCa)&  BCe );

		Ebu ^= Du;
		BCa = ROL(Ebu, 27);
		Ega ^= Da;
		BCe = ROL(Ega, 36);
		Eke ^= De;
		BCi = ROL(Eke, 10);
		Emi ^= Di;
		BCo = ROL(Emi, 15);
		Eso ^= Do;
		BCu = ROR8(Eso);
		Ama =   BCa ^((~BCe)&  BCi );
		Ame =   BCe ^((~BCi)&  BCo );
		Ami =   BCi ^((~BCo)&  BCu );
		Amo =   BCo ^((~BCu)&  BCa );
		Amu =   BCu ^((~BCa)&  BCe );

		Ebi ^= Di;
		BCa = ROL(Ebi, 62);
		Ego ^= Do;
		BCe = ROL(Ego, 55);
		Eku ^= Du;
		BCi = ROL(Eku, 39);
		Ema ^= Da;
		BCo = ROL(Ema, 41);
		Ese ^= De;
		BCu = ROL(Ese, 2);
		Asa =   BCa ^((~BCe)&  BCi );
		Ase =   BCe ^((~BCi)&  BCo );
		Asi =   BCi ^((~BCo)&  BCu );
		Aso =   BCo ^((~BCu)&  BCa );
		Asu =   BCu ^((~BCa)&  BCe );
	}

	//    prepareTheta
	BCa = Aba^Aga^Aka^Ama^Asa;
	BCe = Abe^Age^Ake^Ame^Ase;
	BCi = Abi^Agi^Aki^Ami^Asi;
	BCo = Abo^Ago^Ako^Amo^Aso;
	BCu = Abu^Agu^Aku^Amu^Asu;

	//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
	Da = BCu^ROL(BCe, 1);
	De = BCa^ROL(BCi, 1);
	Di = BCe^ROL(BCo, 1);
	Do = BCi^ROL(BCu, 1);
	Du = BCo^ROL(BCa, 1);

	Aba ^= Da;
	BCa = Aba;
	Age ^= De;
	BCe = ROL(Age, 44);
	Aki ^= Di;
	BCi = ROL(Aki, 43);
	Amo ^= Do;
	BCo = ROL(Amo, 21);
	Asu ^= Du;
	BCu = ROL(Asu, 14);
	Eba =   BCa ^((~BCe)&  BCi );
	Eba ^= vectorize((uint64_t)KeccakF_RoundConstants[22]);
	Ebe =   BCe ^((~BCi)&  BCo );
	Ebi =   BCi ^((~BCo)&  BCu );
	Ebo =   BCo ^((~BCu)&  BCa );
	Ebu =   BCu ^((~BCa)&  BCe );

	Abo ^= Do;
	BCa = ROL(Abo, 28);
	Agu ^= Du;
	BCe = ROL(Agu, 20);
	Aka ^= Da;
	BCi = ROL(Aka,  3);
	Ame ^= De;
	BCo = ROL(Ame, 45);
	Asi ^= Di;
	BCu = ROL(Asi, 61);
	Ega =   BCa ^((~BCe)&  BCi );
	Ege =   BCe ^((~BCi)&  BCo );
	Egi =   BCi ^((~BCo)&  BCu );
	Ego =   BCo ^((~BCu)&  BCa );
	Egu =   BCu ^((~BCa)&  BCe );

	Abe ^= De;
	BCa = ROL(Abe,  1);
	Agi ^= Di;
	BCe = ROL(Agi,  6);
	Ako ^= Do;
	BCi = ROL(Ako, 25);
	Amu ^= Du;
	BCo = ROL(Amu,  8);
	Asa ^= Da;
	BCu = ROL(Asa, 18);
	Eka =   BCa ^((~BCe)&  BCi );
	Eke =   BCe ^((~BCi)&  BCo );
	Eki =   BCi ^((~BCo)&  BCu );
	Eko =   BCo ^((~BCu)&  BCa );
	Eku =   BCu ^((~BCa)&  BCe );

	Abu ^= Du;
	BCa = ROL(Abu, 27);
	Aga ^= Da;
	BCe = ROL(Aga, 36);
	Ake ^= De;
	BCi = ROL(Ake, 10);
	Ami ^= Di;
	BCo = ROL(Ami, 15);
	Aso ^= Do;
	BCu = ROR8(Aso);
	Ema =   BCa ^((~BCe)&  BCi );
	Eme =   BCe ^((~BCi)&  BCo );
	Emi =   BCi ^((~BCo)&  BCu );
	Emo =   BCo ^((~BCu)&  BCa );
	Emu =   BCu ^((~BCa)&  BCe );

	Abi ^= Di;
	BCa = ROL(Abi, 62);
	Ago ^= Do;
	BCe = ROL(Ago, 55);
	Aku ^= Du;
	BCi = ROL(Aku, 39);
	Ama ^= Da;
	BCo = ROL(Ama, 41);
	Ase ^= De;
	BCu = ROL(Ase,  2);
	Esa =   BCa ^((~BCe)&  BCi );
	Ese =   BCe ^((~BCi)&  BCo );
	Esi =   BCi ^((~BCo)&  BCu );
	Eso =   BCo ^((~BCu)&  BCa );
	Esu =   BCu ^((~BCa)&  BCe );

	//    prepareTheta
	BCa = Eba^Ega^Eka^Ema^Esa;
	BCe = Ebe^Ege^Eke^Eme^Ese;
	BCi = Ebi^Egi^Eki^Emi^Esi;
	BCo = Ebo^Ego^Eko^Emo^Eso;
	BCu = Ebu^Egu^Eku^Emu^Esu;

	//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
	Da = BCu^ROL(BCe, 1);
	De = BCa^ROL(BCi, 1);
	Di = BCe^ROL(BCo, 1);
	Do = BCi^ROL(BCu, 1);
	Du = BCo^ROL(BCa, 1);

	Eba ^= Da;
	BCa = Eba;
	Ege ^= De;
	BCe = ROL(Ege, 44);
	Eki ^= Di;
	BCi = ROL(Eki, 43);
	Emo ^= Do;
	BCo = ROL(Emo, 21);
	Esu ^= Du;
	BCu = ROL(Esu, 14);
	Aba =   BCa ^((~BCe)&  BCi );
	Aba ^= vectorize((uint64_t)KeccakF_RoundConstants[23]);
	Abe =   BCe ^((~BCi)&  BCo );
	Abi =   BCi ^((~BCo)&  BCu );
	Abo =   BCo ^((~BCu)&  BCa );

	if (validate) {
		g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
		g_out[3] = devectorize(Abo);
		g_out[2] = devectorize(Abi);
		g_out[1] = devectorize(Abe);
		g_out[0] = devectorize(Aba);
	}

	// the likelyhood of meeting the hashing target is so low, that we're not guarding this
	// with atomic writes, locks or similar...
	uint64_t *g_good64 = (uint64_t*)g_good;
	if (devectorize(Abo) <=  ptarget64[3]) {
		if (devectorize(Abo) < g_good64[3]) {
			g_good64[3] = devectorize(Abo);
			g_good64[2] = devectorize(Abi);
			g_good64[1] = devectorize(Abe);
			g_good64[0] = devectorize(Aba);
			g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
		}
	}
}

static std::map<int, uint32_t *> context_good[2];

bool NV2Kernel::prepare_keccak256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
	static bool init[MAX_DEVICES] = {false};
	if (!init[thr_id])
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(KeccakF_RoundConstants), host_KeccakF_RoundConstants, sizeof(host_KeccakF_RoundConstants), 0, hipMemcpyHostToDevice));

		// allocate pinned host memory for good hashes
		uint32_t *tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata64), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

	return context_good[0][thr_id] && context_good[1][thr_id];
}

void NV2Kernel::do_keccak256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
	checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

	titan_crypto_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

	// copy hashes from device memory to host (ALL hashes, lots of data...)
	if (do_d2h && hash != NULL) {
		size_t mem_size = throughput * sizeof(uint32_t) * 8;
		checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
	else if (hash != NULL) {
		// asynchronous copy of winning nonce (just 4 bytes...)
		checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
}


//
// Blakecoin related Keccak implementation (Keccak256)
//

typedef uint32_t sph_u32;
//#define SPH_C32(x) ((sph_u32)(x))
//#define SPH_T32(x) ((x) & SPH_C32(0xFFFFFFFF))
#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define SPH_ROTL32(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
	// Kepler (Compute 3.5)
	#define SPH_ROTL32(a, b) __funnelshift_l( a, a, b );
#endif
#define SPH_ROTR32(x, n)   SPH_ROTL32(x, (32 - (n)))

__constant__ uint32_t pdata[20];

#ifdef _MSC_VER
#pragma warning (disable: 4146)
#endif

static __device__ __forceinline__ sph_u32 cuda_sph_bswap32(sph_u32 x)
{
	return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
		  | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

/**
 * Encode a 32-bit value into the provided buffer (big endian convention).
 *
 * @param dst   the destination buffer
 * @param val   the 32-bit value to encode
 */
static __device__ __forceinline__ void
cuda_sph_enc32be(void *dst, sph_u32 val)
{
	*(sph_u32 *)dst = cuda_sph_bswap32(val);
}

#define Z00   0
#define Z01   1
#define Z02   2
#define Z03   3
#define Z04   4
#define Z05   5
#define Z06   6
#define Z07   7
#define Z08   8
#define Z09   9
#define Z0A   A
#define Z0B   B
#define Z0C   C
#define Z0D   D
#define Z0E   E
#define Z0F   F

#define Z10   E
#define Z11   A
#define Z12   4
#define Z13   8
#define Z14   9
#define Z15   F
#define Z16   D
#define Z17   6
#define Z18   1
#define Z19   C
#define Z1A   0
#define Z1B   2
#define Z1C   B
#define Z1D   7
#define Z1E   5
#define Z1F   3

#define Z20   B
#define Z21   8
#define Z22   C
#define Z23   0
#define Z24   5
#define Z25   2
#define Z26   F
#define Z27   D
#define Z28   A
#define Z29   E
#define Z2A   3
#define Z2B   6
#define Z2C   7
#define Z2D   1
#define Z2E   9
#define Z2F   4

#define Z30   7
#define Z31   9
#define Z32   3
#define Z33   1
#define Z34   D
#define Z35   C
#define Z36   B
#define Z37   E
#define Z38   2
#define Z39   6
#define Z3A   5
#define Z3B   A
#define Z3C   4
#define Z3D   0
#define Z3E   F
#define Z3F   8

#define Z40   9
#define Z41   0
#define Z42   5
#define Z43   7
#define Z44   2
#define Z45   4
#define Z46   A
#define Z47   F
#define Z48   E
#define Z49   1
#define Z4A   B
#define Z4B   C
#define Z4C   6
#define Z4D   8
#define Z4E   3
#define Z4F   D

#define Z50   2
#define Z51   C
#define Z52   6
#define Z53   A
#define Z54   0
#define Z55   B
#define Z56   8
#define Z57   3
#define Z58   4
#define Z59   D
#define Z5A   7
#define Z5B   5
#define Z5C   F
#define Z5D   E
#define Z5E   1
#define Z5F   9

#define Z60   C
#define Z61   5
#define Z62   1
#define Z63   F
#define Z64   E
#define Z65   D
#define Z66   4
#define Z67   A
#define Z68   0
#define Z69   7
#define Z6A   6
#define Z6B   3
#define Z6C   9
#define Z6D   2
#define Z6E   8
#define Z6F   B

#define Z70   D
#define Z71   B
#define Z72   7
#define Z73   E
#define Z74   C
#define Z75   1
#define Z76   3
#define Z77   9
#define Z78   5
#define Z79   0
#define Z7A   F
#define Z7B   4
#define Z7C   8
#define Z7D   6
#define Z7E   2
#define Z7F   A

#define Z80   6
#define Z81   F
#define Z82   E
#define Z83   9
#define Z84   B
#define Z85   3
#define Z86   0
#define Z87   8
#define Z88   C
#define Z89   2
#define Z8A   D
#define Z8B   7
#define Z8C   1
#define Z8D   4
#define Z8E   A
#define Z8F   5

#define Z90   A
#define Z91   2
#define Z92   8
#define Z93   4
#define Z94   7
#define Z95   6
#define Z96   1
#define Z97   5
#define Z98   F
#define Z99   B
#define Z9A   9
#define Z9B   E
#define Z9C   3
#define Z9D   C
#define Z9E   D
#define Z9F   0

#define Mx(r, i)    Mx_(Z ## r ## i)
#define Mx_(n)      Mx__(n)
#define Mx__(n)     M ## n

#define CSx(r, i)   CSx_(Z ## r ## i)
#define CSx_(n)     CSx__(n)
#define CSx__(n)    CS ## n

#define CS0   SPH_C32(0x243F6A88)
#define CS1   SPH_C32(0x85A308D3)
#define CS2   SPH_C32(0x13198A2E)
#define CS3   SPH_C32(0x03707344)
#define CS4   SPH_C32(0xA4093822)
#define CS5   SPH_C32(0x299F31D0)
#define CS6   SPH_C32(0x082EFA98)
#define CS7   SPH_C32(0xEC4E6C89)
#define CS8   SPH_C32(0x452821E6)
#define CS9   SPH_C32(0x38D01377)
#define CSA   SPH_C32(0xBE5466CF)
#define CSB   SPH_C32(0x34E90C6C)
#define CSC   SPH_C32(0xC0AC29B7)
#define CSD   SPH_C32(0xC97C50DD)
#define CSE   SPH_C32(0x3F84D5B5)
#define CSF   SPH_C32(0xB5470917)

#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
		GS(Mx(r, 0), Mx(r, 1), CSx(r, 0), CSx(r, 1), V0, V4, V8, VC); \
		GS(Mx(r, 2), Mx(r, 3), CSx(r, 2), CSx(r, 3), V1, V5, V9, VD); \
		GS(Mx(r, 4), Mx(r, 5), CSx(r, 4), CSx(r, 5), V2, V6, VA, VE); \
		GS(Mx(r, 6), Mx(r, 7), CSx(r, 6), CSx(r, 7), V3, V7, VB, VF); \
		GS(Mx(r, 8), Mx(r, 9), CSx(r, 8), CSx(r, 9), V0, V5, VA, VF); \
		GS(Mx(r, A), Mx(r, B), CSx(r, A), CSx(r, B), V1, V6, VB, VC); \
		GS(Mx(r, C), Mx(r, D), CSx(r, C), CSx(r, D), V2, V7, V8, VD); \
		GS(Mx(r, E), Mx(r, F), CSx(r, E), CSx(r, F), V3, V4, V9, VE); \
	} while (0)

#define COMPRESS32   do { \
		sph_u32 M0, M1, M2, M3, M4, M5, M6, M7; \
		sph_u32 M8, M9, MA, MB, MC, MD, ME, MF; \
		sph_u32 V0, V1, V2, V3, V4, V5, V6, V7; \
		sph_u32 V8, V9, VA, VB, VC, VD, VE, VF; \
		V0 = H0; \
		V1 = H1; \
		V2 = H2; \
		V3 = H3; \
		V4 = H4; \
		V5 = H5; \
		V6 = H6; \
		V7 = H7; \
		V8 = S0 ^ CS0; \
		V9 = S1 ^ CS1; \
		VA = S2 ^ CS2; \
		VB = S3 ^ CS3; \
		VC = T0 ^ CS4; \
		VD = T0 ^ CS5; \
		VE = T1 ^ CS6; \
		VF = T1 ^ CS7; \
		M0 = input[0]; \
		M1 = input[1]; \
		M2 = input[2]; \
		M3 = input[3]; \
		M4 = input[4]; \
		M5 = input[5]; \
		M6 = input[6]; \
		M7 = input[7]; \
		M8 = input[8]; \
		M9 = input[9]; \
		MA = input[10]; \
		MB = input[11]; \
		MC = input[12]; \
		MD = input[13]; \
		ME = input[14]; \
		MF = input[15]; \
		ROUND_S(0); \
		ROUND_S(1); \
		ROUND_S(2); \
		ROUND_S(3); \
		ROUND_S(4); \
		ROUND_S(5); \
		ROUND_S(6); \
		ROUND_S(7); \
		H0 ^= S0 ^ V0 ^ V8; \
		H1 ^= S1 ^ V1 ^ V9; \
		H2 ^= S2 ^ V2 ^ VA; \
		H3 ^= S3 ^ V3 ^ VB; \
		H4 ^= S0 ^ V4 ^ VC; \
		H5 ^= S1 ^ V5 ^ VD; \
		H6 ^= S2 ^ V6 ^ VE; \
		H7 ^= S3 ^ V7 ^ VF; \
	} while (0)


__global__ void titan_blake256_hash(uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate)
{
	uint32_t input[16];
	uint64_t output[4];

#pragma unroll 16
	for (int i=0; i < 16; ++i) input[i] = pdata[i];

	sph_u32 H0 = 0x6A09E667;
	sph_u32 H1 = 0xBB67AE85;
	sph_u32 H2 = 0x3C6EF372;
	sph_u32 H3 = 0xA54FF53A;
	sph_u32 H4 = 0x510E527F;
	sph_u32 H5 = 0x9B05688C;
	sph_u32 H6 = 0x1F83D9AB;
	sph_u32 H7 = 0x5BE0CD19;
	sph_u32 S0 = 0;
	sph_u32 S1 = 0;
	sph_u32 S2 = 0;
	sph_u32 S3 = 0;
	sph_u32 T0 = 0;
	sph_u32 T1 = 0;
	T0 = SPH_T32(T0 + 512);
	COMPRESS32;

#pragma unroll 3
	for (int i=0; i < 3; ++i) input[i] = pdata[16+i];
	input[3] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
	input[4] = 0x80000000;
#pragma unroll 8
	for (int i=5; i < 13; ++i) input[i] = 0;
	input[13] = 0x00000001;
	input[14] = T1;
	input[15] = T0 + 128;

	T0 = SPH_T32(T0 + 128);
	COMPRESS32;

	cuda_sph_enc32be((unsigned char*)output + 4*6, H6);
	cuda_sph_enc32be((unsigned char*)output + 4*7, H7);
	if (validate || output[3] <=  ptarget64[3])
	{
		// this data is only needed when we actually need to save the hashes
		cuda_sph_enc32be((unsigned char*)output + 4*0, H0);
		cuda_sph_enc32be((unsigned char*)output + 4*1, H1);
		cuda_sph_enc32be((unsigned char*)output + 4*2, H2);
		cuda_sph_enc32be((unsigned char*)output + 4*3, H3);
		cuda_sph_enc32be((unsigned char*)output + 4*4, H4);
		cuda_sph_enc32be((unsigned char*)output + 4*5, H5);
	}

	if (validate)
	{
		g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
#pragma unroll 4
		for (int i=0; i < 4; ++i) g_out[i] = output[i];
	}

	if (output[3] <=  ptarget64[3]) {
		uint64_t *g_good64 = (uint64_t*)g_good;
		if (output[3] < g_good64[3]) {
			g_good64[3] = output[3];
			g_good64[2] = output[2];
			g_good64[1] = output[1];
			g_good64[0] = output[0];
			g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
		}
	}
}

bool NV2Kernel::prepare_blake256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
	static bool init[MAX_DEVICES] = {false};
	if (!init[thr_id])
	{
		// allocate pinned host memory for good hashes
		uint32_t *tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

	return context_good[0][thr_id] && context_good[1][thr_id];
}

void NV2Kernel::do_blake256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
	checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

	titan_blake256_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

	// copy hashes from device memory to host (ALL hashes, lots of data...)
	if (do_d2h && hash != NULL) {
		size_t mem_size = throughput * sizeof(uint32_t) * 8;
		checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
	else if (hash != NULL) {
		// asynchronous copy of winning nonce (just 4 bytes...)
		checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
}
