#include "hip/hip_runtime.h"
//
// Experimental Kernel for Kepler (Compute 3.5) devices
// code submitted by nVidia performance engineer Alexey Panteleev
// with modifications by Christian Buchner
//
// for Compute 3.5
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=80
// for Compute 3.0
// NOTE: compile this .cu module for compute_30,sm_30 with --maxrregcount=63
//

#include <map>

#include "hip/hip_runtime.h"

#include "miner.h"
#include "salsa_kernel.h"
#include "nv_kernel.h"

#define THREADS_PER_WU 1  // single thread per hash

#define TEXWIDTH 32768

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define __ldg(x) (*(x))
#endif

// grab lane ID
static __device__ __inline__ unsigned int __laneId() { unsigned int laneId; asm( "mov.u32 %0, %%laneid;" : "=r"( laneId ) ); return laneId; }

// forward references
template <int ALGO> __global__ void nv_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end);
template <int ALGO, int TEX_DIM> __global__ void nv_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end);
template <int ALGO> __global__ void nv_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP);
template <int ALGO, int TEX_DIM> __global__ void nv_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1; // N - 1
__constant__ uint32_t c_spacing; // (N+LOOKUP_GAP-1)/LOOKUP_GAP

NVKernel::NVKernel() : KernelInterface()
{
}

bool NVKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef1D_4_V.normalized = 0;
	texRef1D_4_V.filterMode = hipFilterModePoint;
	texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
	checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
	return true;
}

bool NVKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef2D_4_V.normalized = 0;
	texRef2D_4_V.filterMode = hipFilterModePoint;
	texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
	texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
	// maintain texture width of TEXWIDTH (max. limit is 65000)
	while (width > TEXWIDTH) { width /= 2; height *= 2; pitch /= 2; }
	while (width < TEXWIDTH) { width *= 2; height = (height+1)/2; pitch *= 2; }
	checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
	return true;
}

bool NVKernel::unbindtexture_1D()
{
	checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
	return true;
}

bool NVKernel::unbindtexture_2D()
{
	checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
	return true;
}

void NVKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool NVKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;

	// make some constants available to kernel, update only initially and when changing
	static int prev_N[MAX_DEVICES] = {0};
	if (N != prev_N[thr_id]) {
		uint32_t h_N = N;
		uint32_t h_N_1 = N-1;
		uint32_t h_spacing = (N+LOOKUP_GAP-1)/LOOKUP_GAP;

		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_spacing), &h_spacing, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);

		prev_N[thr_id] = N;
	}

	// First phase: Sequential writes to scratchpad.
	const int batch = device_batchsize[thr_id];
	unsigned int pos = 0;

	do
	{
		if (LOOKUP_GAP == 1) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelA<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelA<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
			}
		else {
				if (IS_SCRYPT())      nv_scrypt_core_kernelA_LG<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelA_LG<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
			}

		pos += batch;
	} while (pos < N);

	// Second phase: Random read access from scratchpad.
	pos = 0;
	do
	{
		if (LOOKUP_GAP == 1) {
			if (texture_cache == 0) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB<A_SCRYPT     ,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB<A_SCRYPT_JANE,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			}
			else if (texture_cache == 1) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB<A_SCRYPT     ,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB<A_SCRYPT_JANE,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			}
			else if (texture_cache == 2) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB<A_SCRYPT     ,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB<A_SCRYPT_JANE,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			}
		} else {
			if (texture_cache == 0) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB_LG<A_SCRYPT     ,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB_LG<A_SCRYPT_JANE,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
			else if (texture_cache == 1) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB_LG<A_SCRYPT     ,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB_LG<A_SCRYPT_JANE,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
			else if (texture_cache == 2) {
				if (IS_SCRYPT())      nv_scrypt_core_kernelB_LG<A_SCRYPT     ,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) nv_scrypt_core_kernelB_LG<A_SCRYPT_JANE,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
		}

		pos += batch;
	} while (pos < N);

	return success;
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
	left.x ^= right.x;
	left.y ^= right.y;
	left.z ^= right.z;
	left.w ^= right.w;
	return left;
}

__device__ __forceinline__ uint4 __shfl(const uint4 val, unsigned int lane, unsigned int width)
{
	return make_uint4(
		(unsigned int)__shfl((int)val.x, lane, width),
		(unsigned int)__shfl((int)val.y, lane, width),
		(unsigned int)__shfl((int)val.z, lane, width),
		(unsigned int)__shfl((int)val.w, lane, width));
}

__device__ __forceinline__ void __transposed_write_BC(uint4 (&B)[4], uint4 (&C)[4], uint4 *D, int spacing)
{
	unsigned int laneId = __laneId();

	unsigned int lane8 = laneId%8;
	unsigned int tile  = laneId/8;

	uint4 T1[8], T2[8];

	/* Source matrix, A-H are threads, 0-7 are data items, thread A is marked with `*`:

	   *A0  B0  C0  D0  E0  F0  G0  H0
	   *A1  B1  C1  D1  E1  F1  G1  H1
	   *A2  B2  C2  D2  E2  F2  G2  H2
	   *A3  B3  C3  D3  E3  F3  G3  H3
	   *A4  B4  C4  D4  E4  F4  G4  H4
	   *A5  B5  C5  D5  E5  F5  G5  H5
	   *A6  B6  C6  D6  E6  F6  G6  H6
	   *A7  B7  C7  D7  E7  F7  G7  H7
	*/

	// rotate rows
	T1[0] = B[0];
	T1[1] = __shfl(B[1], lane8 + 7, 8);
	T1[2] = __shfl(B[2], lane8 + 6, 8);
	T1[3] = __shfl(B[3], lane8 + 5, 8);
	T1[4] = __shfl(C[0], lane8 + 4, 8);
	T1[5] = __shfl(C[1], lane8 + 3, 8);
	T1[6] = __shfl(C[2], lane8 + 2, 8);
	T1[7] = __shfl(C[3], lane8 + 1, 8);

	/* Matrix after row rotates:

	   *A0  B0  C0  D0  E0  F0  G0  H0
		H1 *A1  B1  C1  D1  E1  F1  G1
		G2  H2 *A2  B2  C2  D2  E2  F2
		F3  G3  H3 *A3  B3  C3  D3  E3
		E4  F4  G4  H4 *A4  B4  C4  D4
		D5  E5  F5  G5  H5 *A5  B5  C5
		C6  D6  E6  F6  G6  H6 *A6  B6
		B7  C7  D7  E7  F7  G7  H7 *A7
	*/

	// rotate columns up using a barrel shifter simulation
	// column X is rotated up by (X+1) items
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T1[n] = ((lane8+1) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 4) ? T1[(n+4) % 8] : T1[n];

	/* Matrix after column rotates:

		H1  H2  H3  H4  H5  H6  H7  H0
		G2  G3  G4  G5  G6  G7  G0  G1
		F3  F4  F5  F6  F7  F0  F1  F2
		E4  E5  E6  E7  E0  E1  E2  E3
		D5  D6  D7  D0  D1  D2  D3  D4
		C6  C7  C0  C1  C2  C3  C4  C5
		B7  B0  B1  B2  B3  B4  B5  B6
	   *A0 *A1 *A2 *A3 *A4 *A5 *A6 *A7
	*/

	// rotate rows again using address math and write to D, in reverse row order
	D[spacing*2*(32*tile   )+ lane8     ] = T2[7];
	D[spacing*2*(32*tile+4 )+(lane8+7)%8] = T2[6];
	D[spacing*2*(32*tile+8 )+(lane8+6)%8] = T2[5];
	D[spacing*2*(32*tile+12)+(lane8+5)%8] = T2[4];
	D[spacing*2*(32*tile+16)+(lane8+4)%8] = T2[3];
	D[spacing*2*(32*tile+20)+(lane8+3)%8] = T2[2];
	D[spacing*2*(32*tile+24)+(lane8+2)%8] = T2[1];
	D[spacing*2*(32*tile+28)+(lane8+1)%8] = T2[0];
}

template <int TEX_DIM> __device__ __forceinline__ void __transposed_read_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	unsigned int laneId = __laneId();

	unsigned int lane8 = laneId%8;
	unsigned int tile  = laneId/8;

	// Perform the same transposition as in __transposed_write_BC, but in reverse order.
	// See the illustrations in comments for __transposed_write_BC.

	// read and rotate rows, in reverse row order
	uint4 T1[8], T2[8];
	const uint4 *loc;
	loc = &S[(spacing*2*(32*tile   ) +  lane8      + 8*__shfl(row, 0, 8))];
	T1[7] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+4 ) + (lane8+7)%8 + 8*__shfl(row, 1, 8))];
	T1[6] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+8 ) + (lane8+6)%8 + 8*__shfl(row, 2, 8))];
	T1[5] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+12) + (lane8+5)%8 + 8*__shfl(row, 3, 8))];
	T1[4] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+16) + (lane8+4)%8 + 8*__shfl(row, 4, 8))];
	T1[3] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+20) + (lane8+3)%8 + 8*__shfl(row, 5, 8))];
	T1[2] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+24) + (lane8+2)%8 + 8*__shfl(row, 6, 8))];
	T1[1] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));
	loc = &S[(spacing*2*(32*tile+28) + (lane8+1)%8 + 8*__shfl(row, 7, 8))];
	T1[0] = TEX_DIM==0 ? __ldg(loc) : TEX_DIM==1 ? tex1Dfetch(texRef1D_4_V, loc-(uint4*)c_V[0]) : tex2D(texRef2D_4_V, 0.5f + ((loc-(uint4*)c_V[0])%TEXWIDTH), 0.5f + ((loc-(uint4*)c_V[0])/TEXWIDTH));

	// rotate columns down using a barrel shifter simulation
	// column X is rotated down by (X+1) items, or up by (8-(X+1)) = (7-X) items
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T1[n] = ((7-lane8) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 4) ? T1[(n+4) % 8] : T1[n];

	// rotate rows
	B[0] = T2[0];
	B[1] = __shfl(T2[1], lane8 + 1, 8);
	B[2] = __shfl(T2[2], lane8 + 2, 8);
	B[3] = __shfl(T2[3], lane8 + 3, 8);
	C[0] = __shfl(T2[4], lane8 + 4, 8);
	C[1] = __shfl(T2[5], lane8 + 5, 8);
	C[2] = __shfl(T2[6], lane8 + 6, 8);
	C[3] = __shfl(T2[7], lane8 + 7, 8);

}

template <int TEX_DIM> __device__ __forceinline__ void __transposed_xor_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	uint4 BT[4], CT[4];
	__transposed_read_BC<TEX_DIM>(S, BT, CT, spacing, row);

#pragma unroll 4
	for(int n = 0; n < 4; n++)
	{
		B[n] ^= BT[n];
		C[n] ^= CT[n];
	}
}

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define ROTL(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
	// Kepler (Compute 3.5)
	#define ROTL(a, b) __funnelshift_l( a, a, b );
#endif



#if 0

#define QUARTER(a,b,c,d) \
	a += b; d ^= a; d = ROTL(d,16); \
	c += d; b ^= c; b = ROTL(b,12); \
	a += b; d ^= a; d = ROTL(d,8); \
	c += d; b ^= c; b = ROTL(b,7);

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#else

#define ADD4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 += s1; d2 += s2; d3 += s3; d4 += s4;

#define XOR4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 ^= s1; d2 ^= s2; d3 ^= s3; d4 ^= s4;

#define ROTL4(d1,d2,d3,d4,amt) \
	d1 = ROTL(d1, amt); d2 = ROTL(d2, amt); d3 = ROTL(d3, amt); d4 = ROTL(d4, amt);

#define QROUND(a1,a2,a3,a4, b1,b2,b3,b4, c1,c2,c3,c4, amt) \
	ADD4 (a1,a2,a3,a4, c1,c2,c3,c4) \
	XOR4 (b1,b2,b3,b4, a1,a2,a3,a4) \
	ROTL4(b1,b2,b3,b4, amt)

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#endif


#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 7); a1^=ROTL(a10, 7); a2^=ROTL(a20, 7); a3^=ROTL(a30, 7);\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 9); a1^=ROTL(a10, 9); a2^=ROTL(a20, 9); a3^=ROTL(a30, 9);\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 13); a1^=ROTL(a10, 13); a2^=ROTL(a20, 13); a3^=ROTL(a30, 13);\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 18); a1^=ROTL(a10, 18); a2^=ROTL(a20, 18); a3^=ROTL(a30, 18);\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}


template <int ALGO> static __device__ void block_mixer(uint4 *B, uint4 *C)
{
	switch (ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); break;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Experimental Scrypt core kernel for Kepler devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int ALGO> __global__
void 












































































nv_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC<0>((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_N);
		++i;
	} else
		__transposed_read_BC<0>((uint4*)(V + (i-1)*32), B, C, c_N, 0);

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		__transposed_write_BC(B, C, (uint4*)(V + i*32), c_N);
		++i;
	}
}

template <int ALGO> __global__
void nv_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC<0>((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_spacing);
		++i;
	} else {
		int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
		__transposed_read_BC<0>((uint4*)(V + pos*32), B, C, c_spacing, 0);
		while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	}

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		if (i % LOOKUP_GAP == 0)
		  __transposed_write_BC(B, C, (uint4*)(V + (i/LOOKUP_GAP)*32), c_spacing);
		++i;
	}
}

template <int ALGO, int TEX_DIM>__global__
void nv_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
		__transposed_read_BC<TEX_DIM>((uint4*)V, B, C, c_N, c_N_1);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	} else
		__transposed_read_BC<0>((uint4*)g_odata, B, C, 1, 0);

	for (int i = begin; i < end; i++)  {
		int slot = C[0].x & c_N_1;
		__transposed_xor_BC<TEX_DIM>((uint4*)(V), B, C, c_N, slot);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}

template <int ALGO, int TEX_DIM> __global__
void nv_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
	  int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
	  __transposed_read_BC<TEX_DIM>((uint4*)V, B, C, c_spacing, pos);
	  while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	} else {
		__transposed_read_BC<TEX_DIM>((uint4*)g_odata, B, C, 1, 0);
	}

	for (int i = begin; i < end; i++)  {
		int slot = C[0].x & c_N_1;
		int pos = slot/LOOKUP_GAP, loop = slot-pos*LOOKUP_GAP;
		uint4 b[4], c[4]; __transposed_read_BC<TEX_DIM>((uint4*)(V), b, c, c_spacing, pos);
		while(loop--) { block_mixer<ALGO>(b, c); block_mixer<ALGO>(c, b); }
#pragma unroll 4
		for(int n = 0; n < 4; n++) { B[n] ^= b[n]; C[n] ^= c[n]; }
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}



//
// Maxcoin related Keccak implementation (Keccak256)
//

// from salsa_kernel.cu
extern std::map<int, int> context_blocks;
extern std::map<int, int> context_wpb;
extern std::map<int, KernelInterface *> context_kernel;
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

__constant__ uint64_t ptarget64[4];

#define ROL(a, offset) ((((uint64_t)a) << ((offset) % 64)) ^ (((uint64_t)a) >> (64-((offset) % 64))))
#define ROL_mult8(a, offset) ROL(a, offset)

__constant__ uint64_t KeccakF_RoundConstants[24];
static uint64_t host_KeccakF_RoundConstants[24] = {
	(uint64_t)0x0000000000000001ULL,
	(uint64_t)0x0000000000008082ULL,
	(uint64_t)0x800000000000808aULL,
	(uint64_t)0x8000000080008000ULL,
	(uint64_t)0x000000000000808bULL,
	(uint64_t)0x0000000080000001ULL,
	(uint64_t)0x8000000080008081ULL,
	(uint64_t)0x8000000000008009ULL,
	(uint64_t)0x000000000000008aULL,
	(uint64_t)0x0000000000000088ULL,
	(uint64_t)0x0000000080008009ULL,
	(uint64_t)0x000000008000000aULL,
	(uint64_t)0x000000008000808bULL,
	(uint64_t)0x800000000000008bULL,
	(uint64_t)0x8000000000008089ULL,
	(uint64_t)0x8000000000008003ULL,
	(uint64_t)0x8000000000008002ULL,
	(uint64_t)0x8000000000000080ULL,
	(uint64_t)0x000000000000800aULL,
	(uint64_t)0x800000008000000aULL,
	(uint64_t)0x8000000080008081ULL,
	(uint64_t)0x8000000000008080ULL,
	(uint64_t)0x0000000080000001ULL,
	(uint64_t)0x8000000080008008ULL
};

__constant__ uint64_t pdata64[10];

static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
		  | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__global__
void kepler_crypto_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
	uint64_t Aba, Abe, Abi, Abo, Abu;
	uint64_t Aga, Age, Agi, Ago, Agu;
	uint64_t Aka, Ake, Aki, Ako, Aku;
	uint64_t Ama, Ame, Ami, Amo, Amu;
	uint64_t Asa, Ase, Asi, Aso, Asu;
	uint64_t BCa, BCe, BCi, BCo, BCu;
	uint64_t Da, De, Di, Do, Du;
	uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
	uint64_t Ega, Ege, Egi, Ego, Egu;
	uint64_t Eka, Eke, Eki, Eko, Eku;
	uint64_t Ema, Eme, Emi, Emo, Emu;
	uint64_t Esa, Ese, Esi, Eso, Esu;

	//copyFromState(A, state)
	Aba = pdata64[0];
	Abe = pdata64[1];
	Abi = pdata64[2];
	Abo = pdata64[3];
	Abu = pdata64[4];
	Aga = pdata64[5];
	Age = pdata64[6];
	Agi = pdata64[7];
	Ago = pdata64[8];
	Agu = (pdata64[9] & 0x00000000FFFFFFFFULL) | (((uint64_t)cuda_swab32(nonce + ((blockIdx.x * blockDim.x) + threadIdx.x))) << 32);
	Aka = 0x0000000000000001ULL;
	Ake = 0;
	Aki = 0;
	Ako = 0;
	Aku = 0;
	Ama = 0;
	Ame = 0x8000000000000000ULL;
	Ami = 0;
	Amo = 0;
	Amu = 0;
	Asa = 0;
	Ase = 0;
	Asi = 0;
	Aso = 0;
	Asu = 0;

#pragma unroll 12
	for( int laneCount = 0; laneCount < 24; laneCount += 2 )
	{
		//    prepareTheta
		BCa = Aba^Aga^Aka^Ama^Asa;
		BCe = Abe^Age^Ake^Ame^Ase;
		BCi = Abi^Agi^Aki^Ami^Asi;
		BCo = Abo^Ago^Ako^Amo^Aso;
		BCu = Abu^Agu^Aku^Amu^Asu;

		//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
		Da = BCu^ROL(BCe, 1);
		De = BCa^ROL(BCi, 1);
		Di = BCe^ROL(BCo, 1);
		Do = BCi^ROL(BCu, 1);
		Du = BCo^ROL(BCa, 1);

		Aba ^= Da;
		BCa = Aba;
		Age ^= De;
		BCe = ROL(Age, 44);
		Aki ^= Di;
		BCi = ROL(Aki, 43);
		Amo ^= Do;
		BCo = ROL(Amo, 21);
		Asu ^= Du;
		BCu = ROL(Asu, 14);
		Eba =   BCa ^((~BCe)&  BCi );
		Eba ^= (uint64_t)KeccakF_RoundConstants[laneCount];
		Ebe =   BCe ^((~BCi)&  BCo );
		Ebi =   BCi ^((~BCo)&  BCu );
		Ebo =   BCo ^((~BCu)&  BCa );
		Ebu =   BCu ^((~BCa)&  BCe );

		Abo ^= Do;
		BCa = ROL(Abo, 28);
		Agu ^= Du;
		BCe = ROL(Agu, 20);
		Aka ^= Da;
		BCi = ROL(Aka,  3);
		Ame ^= De;
		BCo = ROL(Ame, 45);
		Asi ^= Di;
		BCu = ROL(Asi, 61);
		Ega =   BCa ^((~BCe)&  BCi );
		Ege =   BCe ^((~BCi)&  BCo );
		Egi =   BCi ^((~BCo)&  BCu );
		Ego =   BCo ^((~BCu)&  BCa );
		Egu =   BCu ^((~BCa)&  BCe );

		Abe ^= De;
		BCa = ROL(Abe,  1);
		Agi ^= Di;
		BCe = ROL(Agi,  6);
		Ako ^= Do;
		BCi = ROL(Ako, 25);
		Amu ^= Du;
		BCo = ROL_mult8(Amu,  8);
		Asa ^= Da;
		BCu = ROL(Asa, 18);
		Eka =   BCa ^((~BCe)&  BCi );
		Eke =   BCe ^((~BCi)&  BCo );
		Eki =   BCi ^((~BCo)&  BCu );
		Eko =   BCo ^((~BCu)&  BCa );
		Eku =   BCu ^((~BCa)&  BCe );

		Abu ^= Du;
		BCa = ROL(Abu, 27);
		Aga ^= Da;
		BCe = ROL(Aga, 36);
		Ake ^= De;
		BCi = ROL(Ake, 10);
		Ami ^= Di;
		BCo = ROL(Ami, 15);
		Aso ^= Do;
		BCu = ROL_mult8(Aso, 56);
		Ema =   BCa ^((~BCe)&  BCi );
		Eme =   BCe ^((~BCi)&  BCo );
		Emi =   BCi ^((~BCo)&  BCu );
		Emo =   BCo ^((~BCu)&  BCa );
		Emu =   BCu ^((~BCa)&  BCe );

		Abi ^= Di;
		BCa = ROL(Abi, 62);
		Ago ^= Do;
		BCe = ROL(Ago, 55);
		Aku ^= Du;
		BCi = ROL(Aku, 39);
		Ama ^= Da;
		BCo = ROL(Ama, 41);
		Ase ^= De;
		BCu = ROL(Ase,  2);
		Esa =   BCa ^((~BCe)&  BCi );
		Ese =   BCe ^((~BCi)&  BCo );
		Esi =   BCi ^((~BCo)&  BCu );
		Eso =   BCo ^((~BCu)&  BCa );
		Esu =   BCu ^((~BCa)&  BCe );

		//    prepareTheta
		BCa = Eba^Ega^Eka^Ema^Esa;
		BCe = Ebe^Ege^Eke^Eme^Ese;
		BCi = Ebi^Egi^Eki^Emi^Esi;
		BCo = Ebo^Ego^Eko^Emo^Eso;
		BCu = Ebu^Egu^Eku^Emu^Esu;

		//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
		Da = BCu^ROL(BCe, 1);
		De = BCa^ROL(BCi, 1);
		Di = BCe^ROL(BCo, 1);
		Do = BCi^ROL(BCu, 1);
		Du = BCo^ROL(BCa, 1);

		Eba ^= Da;
		BCa = Eba;
		Ege ^= De;
		BCe = ROL(Ege, 44);
		Eki ^= Di;
		BCi = ROL(Eki, 43);
		Emo ^= Do;
		BCo = ROL(Emo, 21);
		Esu ^= Du;
		BCu = ROL(Esu, 14);
		Aba =   BCa ^((~BCe)&  BCi );
		Aba ^= (uint64_t)KeccakF_RoundConstants[laneCount+1];
		Abe =   BCe ^((~BCi)&  BCo );
		Abi =   BCi ^((~BCo)&  BCu );
		Abo =   BCo ^((~BCu)&  BCa );
		Abu =   BCu ^((~BCa)&  BCe );

		Ebo ^= Do;
		BCa = ROL(Ebo, 28);
		Egu ^= Du;
		BCe = ROL(Egu, 20);
		Eka ^= Da;
		BCi = ROL(Eka, 3);
		Eme ^= De;
		BCo = ROL(Eme, 45);
		Esi ^= Di;
		BCu = ROL(Esi, 61);
		Aga =   BCa ^((~BCe)&  BCi );
		Age =   BCe ^((~BCi)&  BCo );
		Agi =   BCi ^((~BCo)&  BCu );
		Ago =   BCo ^((~BCu)&  BCa );
		Agu =   BCu ^((~BCa)&  BCe );

		Ebe ^= De;
		BCa = ROL(Ebe, 1);
		Egi ^= Di;
		BCe = ROL(Egi, 6);
		Eko ^= Do;
		BCi = ROL(Eko, 25);
		Emu ^= Du;
		BCo = ROL_mult8(Emu, 8);
		Esa ^= Da;
		BCu = ROL(Esa, 18);
		Aka =   BCa ^((~BCe)&  BCi );
		Ake =   BCe ^((~BCi)&  BCo );
		Aki =   BCi ^((~BCo)&  BCu );
		Ako =   BCo ^((~BCu)&  BCa );
		Aku =   BCu ^((~BCa)&  BCe );

		Ebu ^= Du;
		BCa = ROL(Ebu, 27);
		Ega ^= Da;
		BCe = ROL(Ega, 36);
		Eke ^= De;
		BCi = ROL(Eke, 10);
		Emi ^= Di;
		BCo = ROL(Emi, 15);
		Eso ^= Do;
		BCu = ROL_mult8(Eso, 56);
		Ama =   BCa ^((~BCe)&  BCi );
		Ame =   BCe ^((~BCi)&  BCo );
		Ami =   BCi ^((~BCo)&  BCu );
		Amo =   BCo ^((~BCu)&  BCa );
		Amu =   BCu ^((~BCa)&  BCe );

		Ebi ^= Di;
		BCa = ROL(Ebi, 62);
		Ego ^= Do;
		BCe = ROL(Ego, 55);
		Eku ^= Du;
		BCi = ROL(Eku, 39);
		Ema ^= Da;
		BCo = ROL(Ema, 41);
		Ese ^= De;
		BCu = ROL(Ese, 2);
		Asa =   BCa ^((~BCe)&  BCi );
		Ase =   BCe ^((~BCi)&  BCo );
		Asi =   BCi ^((~BCo)&  BCu );
		Aso =   BCo ^((~BCu)&  BCa );
		Asu =   BCu ^((~BCa)&  BCe );
	}

	if (validate) {
		g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
		g_out[3] = Abo;
		g_out[2] = Abi;
		g_out[1] = Abe;
		g_out[0] = Aba;
	}

	// the likelyhood of meeting the hashing target is so low, that we're not guarding this
	// with atomic writes, locks or similar...
	uint64_t *g_good64 = (uint64_t*)g_good;
	if (Abo <=  ptarget64[3]) {
		if (Abo < g_good64[3]) {
			g_good64[3] = Abo;
			g_good64[2] = Abi;
			g_good64[1] = Abe;
			g_good64[0] = Aba;
			g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
		}
	}
}

static std::map<int, uint32_t *> context_good[2];

bool NVKernel::prepare_keccak256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
	static bool init[MAX_DEVICES] = {false};
	if (!init[thr_id])
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(KeccakF_RoundConstants), host_KeccakF_RoundConstants, sizeof(host_KeccakF_RoundConstants), 0, hipMemcpyHostToDevice));

		// allocate pinned host memory for good hashes
		uint32_t *tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata64), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

	return context_good[0][thr_id] && context_good[1][thr_id];
}

void NVKernel::do_keccak256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
	checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

	kepler_crypto_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

	// copy hashes from device memory to host (ALL hashes, lots of data...)
	if (do_d2h && hash != NULL) {
		size_t mem_size = throughput * sizeof(uint32_t) * 8;
		checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
	else if (hash != NULL) {
		// asynchronous copy of winning nonce (just 4 bytes...)
		checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
}


//
// Blakecoin related Keccak implementation (Keccak256)
//

typedef uint32_t sph_u32;
#define SPH_C32(x) ((sph_u32)(x))
#define SPH_T32(x) ((x) & SPH_C32(0xFFFFFFFF))
#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define SPH_ROTL32(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
	// Kepler (Compute 3.5)
	#define SPH_ROTL32(a, b) __funnelshift_l( a, a, b );
#endif
#define SPH_ROTR32(x, n)   SPH_ROTL32(x, (32 - (n)))

__constant__ uint32_t pdata[20];

#ifdef _MSC_VER
#pragma warning (disable: 4146)
#endif

static __device__ sph_u32 cuda_sph_bswap32(sph_u32 x)
{
	return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
		  | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

/**
 * Encode a 32-bit value into the provided buffer (big endian convention).
 *
 * @param dst   the destination buffer
 * @param val   the 32-bit value to encode
 */
static __device__ void
cuda_sph_enc32be(void *dst, sph_u32 val)
{
	*(sph_u32 *)dst = cuda_sph_bswap32(val);
}

#define Z00   0
#define Z01   1
#define Z02   2
#define Z03   3
#define Z04   4
#define Z05   5
#define Z06   6
#define Z07   7
#define Z08   8
#define Z09   9
#define Z0A   A
#define Z0B   B
#define Z0C   C
#define Z0D   D
#define Z0E   E
#define Z0F   F

#define Z10   E
#define Z11   A
#define Z12   4
#define Z13   8
#define Z14   9
#define Z15   F
#define Z16   D
#define Z17   6
#define Z18   1
#define Z19   C
#define Z1A   0
#define Z1B   2
#define Z1C   B
#define Z1D   7
#define Z1E   5
#define Z1F   3

#define Z20   B
#define Z21   8
#define Z22   C
#define Z23   0
#define Z24   5
#define Z25   2
#define Z26   F
#define Z27   D
#define Z28   A
#define Z29   E
#define Z2A   3
#define Z2B   6
#define Z2C   7
#define Z2D   1
#define Z2E   9
#define Z2F   4

#define Z30   7
#define Z31   9
#define Z32   3
#define Z33   1
#define Z34   D
#define Z35   C
#define Z36   B
#define Z37   E
#define Z38   2
#define Z39   6
#define Z3A   5
#define Z3B   A
#define Z3C   4
#define Z3D   0
#define Z3E   F
#define Z3F   8

#define Z40   9
#define Z41   0
#define Z42   5
#define Z43   7
#define Z44   2
#define Z45   4
#define Z46   A
#define Z47   F
#define Z48   E
#define Z49   1
#define Z4A   B
#define Z4B   C
#define Z4C   6
#define Z4D   8
#define Z4E   3
#define Z4F   D

#define Z50   2
#define Z51   C
#define Z52   6
#define Z53   A
#define Z54   0
#define Z55   B
#define Z56   8
#define Z57   3
#define Z58   4
#define Z59   D
#define Z5A   7
#define Z5B   5
#define Z5C   F
#define Z5D   E
#define Z5E   1
#define Z5F   9

#define Z60   C
#define Z61   5
#define Z62   1
#define Z63   F
#define Z64   E
#define Z65   D
#define Z66   4
#define Z67   A
#define Z68   0
#define Z69   7
#define Z6A   6
#define Z6B   3
#define Z6C   9
#define Z6D   2
#define Z6E   8
#define Z6F   B

#define Z70   D
#define Z71   B
#define Z72   7
#define Z73   E
#define Z74   C
#define Z75   1
#define Z76   3
#define Z77   9
#define Z78   5
#define Z79   0
#define Z7A   F
#define Z7B   4
#define Z7C   8
#define Z7D   6
#define Z7E   2
#define Z7F   A

#define Z80   6
#define Z81   F
#define Z82   E
#define Z83   9
#define Z84   B
#define Z85   3
#define Z86   0
#define Z87   8
#define Z88   C
#define Z89   2
#define Z8A   D
#define Z8B   7
#define Z8C   1
#define Z8D   4
#define Z8E   A
#define Z8F   5

#define Z90   A
#define Z91   2
#define Z92   8
#define Z93   4
#define Z94   7
#define Z95   6
#define Z96   1
#define Z97   5
#define Z98   F
#define Z99   B
#define Z9A   9
#define Z9B   E
#define Z9C   3
#define Z9D   C
#define Z9E   D
#define Z9F   0

#define Mx(r, i)    Mx_(Z ## r ## i)
#define Mx_(n)      Mx__(n)
#define Mx__(n)     M ## n

#define CSx(r, i)   CSx_(Z ## r ## i)
#define CSx_(n)     CSx__(n)
#define CSx__(n)    CS ## n

#define CS0   SPH_C32(0x243F6A88)
#define CS1   SPH_C32(0x85A308D3)
#define CS2   SPH_C32(0x13198A2E)
#define CS3   SPH_C32(0x03707344)
#define CS4   SPH_C32(0xA4093822)
#define CS5   SPH_C32(0x299F31D0)
#define CS6   SPH_C32(0x082EFA98)
#define CS7   SPH_C32(0xEC4E6C89)
#define CS8   SPH_C32(0x452821E6)
#define CS9   SPH_C32(0x38D01377)
#define CSA   SPH_C32(0xBE5466CF)
#define CSB   SPH_C32(0x34E90C6C)
#define CSC   SPH_C32(0xC0AC29B7)
#define CSD   SPH_C32(0xC97C50DD)
#define CSE   SPH_C32(0x3F84D5B5)
#define CSF   SPH_C32(0xB5470917)

#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
		GS(Mx(r, 0), Mx(r, 1), CSx(r, 0), CSx(r, 1), V0, V4, V8, VC); \
		GS(Mx(r, 2), Mx(r, 3), CSx(r, 2), CSx(r, 3), V1, V5, V9, VD); \
		GS(Mx(r, 4), Mx(r, 5), CSx(r, 4), CSx(r, 5), V2, V6, VA, VE); \
		GS(Mx(r, 6), Mx(r, 7), CSx(r, 6), CSx(r, 7), V3, V7, VB, VF); \
		GS(Mx(r, 8), Mx(r, 9), CSx(r, 8), CSx(r, 9), V0, V5, VA, VF); \
		GS(Mx(r, A), Mx(r, B), CSx(r, A), CSx(r, B), V1, V6, VB, VC); \
		GS(Mx(r, C), Mx(r, D), CSx(r, C), CSx(r, D), V2, V7, V8, VD); \
		GS(Mx(r, E), Mx(r, F), CSx(r, E), CSx(r, F), V3, V4, V9, VE); \
	} while (0)

#define COMPRESS32   do { \
		sph_u32 M0, M1, M2, M3, M4, M5, M6, M7; \
		sph_u32 M8, M9, MA, MB, MC, MD, ME, MF; \
		sph_u32 V0, V1, V2, V3, V4, V5, V6, V7; \
		sph_u32 V8, V9, VA, VB, VC, VD, VE, VF; \
		V0 = H0; \
		V1 = H1; \
		V2 = H2; \
		V3 = H3; \
		V4 = H4; \
		V5 = H5; \
		V6 = H6; \
		V7 = H7; \
		V8 = S0 ^ CS0; \
		V9 = S1 ^ CS1; \
		VA = S2 ^ CS2; \
		VB = S3 ^ CS3; \
		VC = T0 ^ CS4; \
		VD = T0 ^ CS5; \
		VE = T1 ^ CS6; \
		VF = T1 ^ CS7; \
		M0 = input[0]; \
		M1 = input[1]; \
		M2 = input[2]; \
		M3 = input[3]; \
		M4 = input[4]; \
		M5 = input[5]; \
		M6 = input[6]; \
		M7 = input[7]; \
		M8 = input[8]; \
		M9 = input[9]; \
		MA = input[10]; \
		MB = input[11]; \
		MC = input[12]; \
		MD = input[13]; \
		ME = input[14]; \
		MF = input[15]; \
		ROUND_S(0); \
		ROUND_S(1); \
		ROUND_S(2); \
		ROUND_S(3); \
		ROUND_S(4); \
		ROUND_S(5); \
		ROUND_S(6); \
		ROUND_S(7); \
		H0 ^= S0 ^ V0 ^ V8; \
		H1 ^= S1 ^ V1 ^ V9; \
		H2 ^= S2 ^ V2 ^ VA; \
		H3 ^= S3 ^ V3 ^ VB; \
		H4 ^= S0 ^ V4 ^ VC; \
		H5 ^= S1 ^ V5 ^ VD; \
		H6 ^= S2 ^ V6 ^ VE; \
		H7 ^= S3 ^ V7 ^ VF; \
	} while (0)


__global__
void kepler_blake256_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate)
{
	uint32_t input[16];
	uint64_t output[4];

	#pragma unroll
	for (int i=0; i < 16; ++i) input[i] = pdata[i];

	sph_u32 H0 = 0x6A09E667;
	sph_u32 H1 = 0xBB67AE85;
	sph_u32 H2 = 0x3C6EF372;
	sph_u32 H3 = 0xA54FF53A;
	sph_u32 H4 = 0x510E527F;
	sph_u32 H5 = 0x9B05688C;
	sph_u32 H6 = 0x1F83D9AB;
	sph_u32 H7 = 0x5BE0CD19;
	sph_u32 S0 = 0;
	sph_u32 S1 = 0;
	sph_u32 S2 = 0;
	sph_u32 S3 = 0;
	sph_u32 T0 = 0;
	sph_u32 T1 = 0;
	T0 = SPH_T32(T0 + 512);
	COMPRESS32;

	#pragma unroll
	for (int i=0; i < 3; ++i) input[i] = pdata[16+i];

	input[3] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
	input[4] = 0x80000000;

	#pragma unroll 8
	for (int i=5; i < 13; ++i) input[i] = 0;

	input[13] = 0x00000001;
	input[14] = T1;
	input[15] = T0 + 128;

	T0 = SPH_T32(T0 + 128);
	COMPRESS32;

	cuda_sph_enc32be((unsigned char*)output + 4*6, H6);
	cuda_sph_enc32be((unsigned char*)output + 4*7, H7);
	if (validate || output[3] <=  ptarget64[3])
	{
		// this data is only needed when we actually need to save the hashes
		cuda_sph_enc32be((unsigned char*)output + 4*0, H0);
		cuda_sph_enc32be((unsigned char*)output + 4*1, H1);
		cuda_sph_enc32be((unsigned char*)output + 4*2, H2);
		cuda_sph_enc32be((unsigned char*)output + 4*3, H3);
		cuda_sph_enc32be((unsigned char*)output + 4*4, H4);
		cuda_sph_enc32be((unsigned char*)output + 4*5, H5);
	}

	if (validate)
	{
		g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
		#pragma unroll
		for (int i=0; i < 4; ++i) g_out[i] = output[i];
	}

	if (output[3] <=  ptarget64[3]) {
		uint64_t *g_good64 = (uint64_t*)g_good;
		if (output[3] < g_good64[3]) {
			g_good64[3] = output[3];
			g_good64[2] = output[2];
			g_good64[1] = output[1];
			g_good64[0] = output[0];
			g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
		}
	}
}

bool NVKernel::prepare_blake256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
	static bool init[MAX_DEVICES] = {false};
	if (!init[thr_id])
	{
		// allocate pinned host memory for good hashes
		uint32_t *tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

	return context_good[0][thr_id] && context_good[1][thr_id];
}

void NVKernel::do_blake256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
	checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

	kepler_blake256_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

	// copy hashes from device memory to host (ALL hashes, lots of data...)
	if (do_d2h && hash != NULL) {
		size_t mem_size = throughput * sizeof(uint32_t) * 8;
		checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
	else if (hash != NULL) {
		// asynchronous copy of winning nonce (just 4 bytes...)
		checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
						hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
	}
}
