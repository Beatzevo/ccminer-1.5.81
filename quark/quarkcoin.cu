#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
uint32_t *d_branch1Nonces[MAX_GPUS];
uint32_t *d_branch2Nonces[MAX_GPUS];
uint32_t *d_branch3Nonces[MAX_GPUS];


extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(uint32_t thr_id, uint64_t *pdata);

extern void quark_blake512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_bmw512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_bmw512_cpu_hash_64_quark(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_groestl512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccakskein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_keccak512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void quark_keccak512_cpu_init(int thr_id);


extern void quark_jh512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_jh512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void quark_jh512_cpu_init(int thr_id);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1,
											uint32_t *d_nonces2, uint32_t *nrm2);
extern void quark_compactTest_single_false_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
	uint32_t *d_nonces1, uint32_t *nrm1);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash);
extern void cuda_check_quarkcoin(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, uint32_t *foundnonces);

// Original Quarkhash Funktion aus einem miner Quelltext
extern "C" void quarkhash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_bmw512_context ctx_bmw;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    unsigned char hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_bmw512_init(&ctx_bmw);
    sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
    sph_bmw512_close(&ctx_bmw, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_groestl512_init(&ctx_groestl);
        sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
        sph_groestl512_close(&ctx_groestl, (void*) hash);
    }
    else
    {
        sph_skein512_init(&ctx_skein);
        sph_skein512 (&ctx_skein, (const void*) hash, 64);
        sph_skein512_close(&ctx_skein, (void*) hash);
    }
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_blake512_init(&ctx_blake);
        sph_blake512 (&ctx_blake, (const void*) hash, 64);
        sph_blake512_close(&ctx_blake, (void*) hash);
    }
    else
    {
        sph_bmw512_init(&ctx_bmw);
        sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
        sph_bmw512_close(&ctx_bmw, (void*) hash);
    }

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_keccak512_init(&ctx_keccak);
        sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
        sph_keccak512_close(&ctx_keccak, (void*) hash);
    }
    else
    {
        sph_jh512_init(&ctx_jh);
        sph_jh512 (&ctx_jh, (const void*) hash, 64);
        sph_jh512_close(&ctx_jh, (void*) hash);
    }

    memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static uint32_t endiandata[MAX_GPUS][20];
static uint32_t foundnonces[MAX_GPUS][2];
static uint32_t foundnonces2[MAX_GPUS][2];

extern "C" int scanhash_quark(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	uint32_t intensity = 256*256*57;
	intensity = intensity + ((1 << 22));
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);

	if (device_sm[device_map[thr_id]] > 500) intensity= 1 << 24;

	if (strstr(props.name, "980 Ti"))
	{
		intensity = 1 << 25;
	} else
	if (strstr(props.name, "980"))
	{
		intensity = 1 << 25;
	}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] =0x2f;

	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		// Konstanten kopieren, Speicher belegen
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		quark_compactTest_cpu_init(thr_id, throughput);

		uint32_t noncebuffersize = throughput * 7 / 10;
		uint32_t noncebuffersize2 = (throughput * 7 / 10)*7/10;

		hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*noncebuffersize2);
		hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*noncebuffersize2);
		hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*noncebuffersize);
		quark_blake512_cpu_init(thr_id);
		quark_keccak512_cpu_init(thr_id);
		quark_jh512_cpu_init(thr_id);
		CUDA_SAFE_CALL(hipGetLastError());
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[thr_id][k], ((uint32_t*)pdata)[k]);
	cuda_check_cpu_setTarget(ptarget);
	if (opt_n_gputhreads > 1)
	{
		quark_blake512_cpu_setBlock_80_multi(thr_id, (uint64_t *)endiandata[thr_id]);
	}
	else
	{
		quark_blake512_cpu_setBlock_80((uint64_t *)endiandata[thr_id]);
	}

	do {

		uint32_t nrm1 = 0, nrm2 = 0, nrm3 = 0;

		quark_blake512_cpu_hash_80( throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64_quark(throughput, pdata[19],d_hash[thr_id]);

		quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
			d_branch3Nonces[thr_id], &nrm3);

		// nur den Skein Branch weiterverfolgen
		quark_skein512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch für Groestl512
		quark_groestl512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch für JH512
		quark_jh512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2);

		// das ist der bedingte Branch für Blake512
		quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id]);

		// das ist der bedingte Branch für Bmw512
		quark_bmw512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id]);

		quark_keccakskein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch3Nonces[thr_id], &nrm2);
		
		quark_keccak512_cpu_hash_64_final(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], ptarget[7], &foundnonces2[thr_id][0]);
		quark_jh512_cpu_hash_64_final(thr_id, nrm2, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], ptarget[7], &foundnonces[thr_id][0]);

		if (foundnonces[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[thr_id][19], foundnonces[thr_id][0]);
			quarkhash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				// check if there was some other ones...
				if (foundnonces2[thr_id][0] != 0xffffffff)
				{
					const uint32_t Htarg = ptarget[7];
					uint32_t vhash64[8];
					be32enc(&endiandata[thr_id][19], foundnonces2[thr_id][0]);
					quarkhash(vhash64, endiandata[thr_id]);

					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundnonces2[thr_id][0];
						res++;
						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces2[thr_id][0]);
					}
					else
					{
						if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
							applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces2[thr_id][0]);
					}
				} else	if (foundnonces[thr_id][1] != 0xffffffff)
				{
					pdata[21] = foundnonces[thr_id][1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces[thr_id][1]);
				}				
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nonce $%08X", thr_id, foundnonces[thr_id][0]);
				pdata[19] = foundnonces[thr_id][0];
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces[thr_id][0]);
			}
		}

		if (foundnonces2[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[thr_id][19], foundnonces2[thr_id][0]);
			quarkhash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				// check if there was some other ones...
				if (foundnonces2[thr_id][1] != 0xffffffff)
				{
					pdata[21] = foundnonces2[thr_id][1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces2[thr_id][1]);
				}
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nonce $%08X", thr_id, foundnonces2[thr_id][0]);
				pdata[19] = foundnonces2[thr_id][0];
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces2[thr_id][0]);
			}
		}


		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
