
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 300
/**
 * __shfl() returns the value of var held by the thread whose ID is given by srcLane.
 * If srcLane is outside the range 0..width-1, the thread's own value of var is returned.
 */
#undef __shfl
#define __shfl(var, srcLane, width) (uint32_t)(var)
#endif

#define merge8(z, x, y, b)\
		z=__byte_perm(x, y, b); \

#define SWAP8(x,y)\
		x=__byte_perm(x, y, 0x5410); \
		y=__byte_perm(x, y, 0x7632);

__device__ __forceinline__ void SWAP4(uint32_t &x, uint32_t &y, uint32_t m)
{
		uint32_t t = (y << 4);	
		t = (x ^ t) & m;
	//	asm("lop3.b32 %0, %1, %2, %3, 0x28;" : "=r"(t) : "r"(x), "r"(t), "r"(m)); //0x28 = (0xF0 ^ 0xCC) & 0xAA 
		x = (x ^ t);
		t = t >> 4;
		y = y ^ t;
}

__device__ __forceinline__ void SWAP2(uint32_t &x, uint32_t &y, uint32_t m)
{
	uint32_t t = (y << 2);
	t = (x ^ t) & m;
	//asm("lop3.b32 %0, %1, %2, %3, 0x28;" : "=r"(t) : "r"(x), "r"(t), "r"(m)); //0x28 = (0xF0 ^ 0xCC) & 0xAA 
	x = (x ^ t);
	t = t >> 2;
	y = y ^ t;
}

__device__ __forceinline__ void SWAP1(uint32_t &x, uint32_t &y, uint32_t m)
{
	uint32_t t = (y << 1);
	t = (x ^ t) & m;
//	asm("lop3.b32 %0, %1, %2, %3, 0x28;" : "=r"(t) : "r"(x), "r"(t), "r"(m)); //0x28 = (0xF0 ^ 0xCC) & 0xAA 
	x = (x ^ t);
	t = t >> 1;
	y = y ^ t;
}

#define SWAP4_final(x,y)\
	asm("and.b32 %0, %0, 0x0f0f0f0f;"\
	 "and.b32 %1, %1, 0x0f0f0f0f;"\
	 "vshl.u32.u32.u32.clamp.add %0, %1, 4, %0;\n\t"\
	: "+r"(x) : "r"(y));\

__device__ __forceinline__
void to_bitslice_quad(uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
    uint32_t other[8];
	uint32_t t;

	uint32_t perm = (threadIdx.x & 1) ? 0x7362 : 0x5140;
	const uint32_t n = threadIdx.x & 3;
#pragma unroll
		for (int i = 0; i < 8; i++)
		{
			input[i] = __shfl((int)input[i], n ^ (3 * (n >= 1 && n <= 2)), 4);
			other[i] = __shfl((int)input[i], (threadIdx.x + 1) & 3, 4);
			input[i] = __shfl((int)input[i], threadIdx.x & 2, 4);
			other[i] = __shfl((int)other[i], threadIdx.x & 2, 4);
		}
		register uint32_t m1 = 0xaaaaaaaaUL;
		register uint32_t m2 = 0xccccccccUL;
		register uint32_t m4 = 0xf0f0f0f0UL;


		merge8(output[0], input[0], input[4], perm);
		merge8(output[1], other[0], other[4], perm);
		merge8(output[2], input[1], input[5], perm);
		merge8(output[3], other[1], other[5], perm);
		merge8(output[4], input[2], input[6], perm);
		merge8(output[5], other[2], other[6], perm);
		merge8(output[6], input[3], input[7], perm);
		merge8(output[7], other[3], other[7], perm);

		SWAP1(output[0], output[1],m1);
		SWAP1(output[2], output[3], m1);
		SWAP1(output[4], output[5], m1);
		SWAP1(output[6], output[7], m1);

		SWAP2(output[0], output[2], m2);
		SWAP2(output[1], output[3], m2);
		SWAP2(output[4], output[6], m2);
		SWAP2(output[5], output[7], m2);

		SWAP4(output[0], output[4], m4);
		SWAP4(output[1], output[5], m4);
		SWAP4(output[2], output[6], m4);
		SWAP4(output[3], output[7], m4);
}

__device__ __forceinline__
void from_bitslice_quad(const uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t t;
	const uint32_t perm = 0x7531;//(threadIdx.x & 1) ? 0x3175 : 0x7531;

	register uint32_t m1 = 0xaaaaaaaaUL;
	register uint32_t m2 = 0xccccccccUL;
	register uint32_t m4 = 0xf0f0f0f0UL;

		output[0] = __byte_perm(input[0], input[4], perm);
		output[2] = __byte_perm(input[1], input[5], perm);
		output[8] = __byte_perm(input[2], input[6], perm);
		output[10] = __byte_perm(input[3], input[7], perm);

		SWAP1(output[0], output[2], m1);
		SWAP1(output[8], output[10], m1);

		SWAP2(output[0], output[8], m2);
		SWAP2(output[2], output[10], m2);

		output[4] = __byte_perm(output[0], output[8], 0x5410);
		output[8] = __byte_perm(output[0], output[8], 0x7632);
		output[0] = output[4];

		output[6] = __byte_perm(output[2], output[10], 0x5410);
		output[10] = __byte_perm(output[2], output[10], 0x7632);
		output[2] = output[6];

		SWAP4(output[0], output[8], m4);
		SWAP4(output[2], output[10], m4);

		if (threadIdx.x & 1)
		{
			output[14] = __byte_perm(output[10], 0, 0x3232);
			output[12] = __byte_perm(output[8], 0, 0x3232);
			output[6] = __byte_perm(output[2], 0, 0x3232);
			output[4] = __byte_perm(output[0], 0, 0x3232);

			output[0] = __byte_perm(output[0], 0, 0x1032);
			output[2] = __byte_perm(output[2], 0, 0x1032);
			output[8] = __byte_perm(output[8], 0, 0x1032);
			output[10] = __byte_perm(output[10], 0, 0x1032);
		}
		else
		{
			output[4] = output[0];
			output[6] = output[2];
			output[12] = output[8];
			output[14] = output[10];
		}

	output[0] = __byte_perm(output[0], __shfl((int)output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[2] = __byte_perm(output[2], __shfl((int)output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[4] = __byte_perm(output[4], __shfl((int)output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[6] = __byte_perm(output[6], __shfl((int)output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[8] = __byte_perm(output[8], __shfl((int)output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[10] = __byte_perm(output[10], __shfl((int)output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[12] = __byte_perm(output[12], __shfl((int)output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[14] = __byte_perm(output[14], __shfl((int)output[14], (threadIdx.x + 1) & 3, 4), 0x7632);

	output[0 + 1] = __shfl((int)output[0], (threadIdx.x + 2) & 3, 4);
	output[2 + 1] = __shfl((int)output[2], (threadIdx.x + 2) & 3, 4);
	output[4 + 1] = __shfl((int)output[4], (threadIdx.x + 2) & 3, 4);
	output[6 + 1] = __shfl((int)output[6], (threadIdx.x + 2) & 3, 4);
	output[8 + 1] = __shfl((int)output[8], (threadIdx.x + 2) & 3, 4);
	output[10 + 1] = __shfl((int)output[10], (threadIdx.x + 2) & 3, 4);
	output[12 + 1] = __shfl((int)output[12], (threadIdx.x + 2) & 3, 4);
	output[14 + 1] = __shfl((int)output[14], (threadIdx.x + 2) & 3, 4);

}

__device__ __forceinline__
void from_bitslice_quad_final(const uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t t;
	const uint32_t perm = 0x7531;//(threadIdx.x & 1) ? 0x3175 : 0x7531;


	register uint32_t m1 = 0xaaaaaaaaUL;
	register uint32_t m2 = 0xccccccccUL;
	register uint32_t m4 = 0xf0f0f0f0UL;

	if (threadIdx.x & 3)
	{

		output[0] = __byte_perm(input[0], input[4], perm);
		output[2] = __byte_perm(input[1], input[5], perm);
		output[8] = __byte_perm(input[2], input[6], perm);
		output[10] = __byte_perm(input[3], input[7], perm);
		SWAP1(output[0], output[2],m1);
		SWAP1(output[8], output[10], m1);
		SWAP2(output[2], output[10], m2);
		output[6] = __byte_perm(output[2], output[10], 0x5410);
		output[10] = __byte_perm(output[2], output[10], 0x7632);
		SWAP4_final(output[6], output[10]);
		output[6] = __byte_perm(output[6], 0, 0x3232);
	} else
	{
		output[0] = __byte_perm(input[0], input[4], perm);
		output[2] = __byte_perm(input[1], input[5], perm);
		output[8] = __byte_perm(input[2], input[6], perm);
		output[10] = __byte_perm(input[3], input[7], perm);

		SWAP1(output[0], output[2],m1);
		SWAP1(output[8], output[10], m1);

		SWAP2(output[0], output[8], m2);
		SWAP2(output[2], output[10], m2);

		output[4] = __byte_perm(output[0], output[8], 0x5410);
		output[8] = __byte_perm(output[0], output[8], 0x7632);
		output[0] = output[4];

		output[6] = __byte_perm(output[2], output[10], 0x5410);
		output[10] = __byte_perm(output[2], output[10], 0x7632);
		output[2] = output[6];

		SWAP4(output[0], output[8], m4);
		SWAP4(output[2], output[10], m4);

		if (threadIdx.x & 1)
		{
			output[14] = __byte_perm(output[10], 0, 0x3232);
			output[12] = __byte_perm(output[8], 0, 0x3232);
			output[6] = __byte_perm(output[2], 0, 0x3232);
			output[4] = __byte_perm(output[0], 0, 0x3232);

			output[0] = __byte_perm(output[0], 0, 0x1032);
			output[2] = __byte_perm(output[2], 0, 0x1032);
			output[8] = __byte_perm(output[8], 0, 0x1032);
			output[10] = __byte_perm(output[10], 0, 0x1032);
		}else
		{
			output[4] = output[0];
			output[6] = output[2];
			output[12] = output[8];
			output[14] = output[10];
		}
	}


	output[0] = __byte_perm(output[0], __shfl((int)output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[2] = __byte_perm(output[2], __shfl((int)output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[4] = __byte_perm(output[4], __shfl((int)output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[6] = __byte_perm(output[6], __shfl((int)output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[8] = __byte_perm(output[8], __shfl((int)output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[10] = __byte_perm(output[10], __shfl((int)output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[12] = __byte_perm(output[12], __shfl((int)output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[14] = __byte_perm(output[14], __shfl((int)output[14], (threadIdx.x + 1) & 3, 4), 0x7632);

	output[0 + 1] = __shfl((int)output[0], (threadIdx.x + 2) & 3, 4);
	output[2 + 1] = __shfl((int)output[2], (threadIdx.x + 2) & 3, 4);
	output[4 + 1] = __shfl((int)output[4], (threadIdx.x + 2) & 3, 4);
	output[6 + 1] = __shfl((int)output[6], (threadIdx.x + 2) & 3, 4);
	output[8 + 1] = __shfl((int)output[8], (threadIdx.x + 2) & 3, 4);
	output[10 + 1] = __shfl((int)output[10], (threadIdx.x + 2) & 3, 4);
	output[12 + 1] = __shfl((int)output[12], (threadIdx.x + 2) & 3, 4);
	output[14 + 1] = __shfl((int)output[14], (threadIdx.x + 2) & 3, 4);

}
