#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include <host_defines.h>

// globaler Speicher für alle HeftyHashes aller Threads
static uint32_t *d_resultNonce[MAX_GPUS];

__constant__ uint32_t groestlcoin_gpu_msg[20];

// 64 Register Variante für Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

#define SWAB32(x) cuda_swab32(x)

__global__ __launch_bounds__(512, 2)
void groestlcoin_gpu_hash_quad(uint32_t threads, uint32_t startNounce, uint32_t *resNounce, uint32_t target)
{
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) / 4;
    if (thread < threads)
    {
        // GROESTL
		uint32_t paddedInput[8] = { 0 };
		uint32_t nounce = startNounce + thread;
		paddedInput[0] = groestlcoin_gpu_msg[(threadIdx.x & 3)];
		paddedInput[1] = groestlcoin_gpu_msg[4 + (threadIdx.x & 3)];
		paddedInput[2] = groestlcoin_gpu_msg[8 + (threadIdx.x & 3)];
		paddedInput[3] = groestlcoin_gpu_msg[12 + (threadIdx.x & 3)];
		paddedInput[4] = groestlcoin_gpu_msg[16 + (threadIdx.x & 3)];
		if ((threadIdx.x & 3) == 3) paddedInput[4] = SWAB32(nounce);
		if ((threadIdx.x & 3) == 0) paddedInput[5] = 0x80;
		if ((threadIdx.x & 3)==3) paddedInput[7] = 0x01000000;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(paddedInput, msgBitsliced);

        uint32_t state[8];

        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + ((threadIdx.x & 3)==3)*0x2000);
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
				msgBitsliced[7] = __byte_perm(state[7], 0x00800100, 0x4341 + ((threadIdx.x & 3) == 0) * 0x0010);
            }
        }

        uint32_t out_state[16];
        from_bitslice_quad_final(state, out_state);
        
		if ((threadIdx.x & 3) == 0)
        {

			if (out_state[7] <= target) 
			{
				atomicCAS(resNounce, 0xffffffff, nounce);
//				if (resNounce[0] > nounce)
//					resNounce[0] = nounce;
			}
        }
    }
}

// Setup-Funktionen
__host__ void groestlcoin_cpu_init(int thr_id, uint32_t threads)
{
    // Speicher für Gewinner-Nonce belegen
    hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
}

__host__ void groestlcoin_cpu_setBlock(int thr_id, void *data )
{
    uint32_t msgBlock[20];
    memcpy(&msgBlock[0], data, 80);
    hipMemcpyToSymbol(HIP_SYMBOL( groestlcoin_gpu_msg),
                        msgBlock,
                        80);

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
}

__host__ void groestlcoin_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce, uint32_t target)
{
    uint32_t threadsperblock = 512;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    int factor = 4;

     // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    hipMemset(d_resultNonce[thr_id], 0xFFFFFFFF, sizeof(uint32_t));
    groestlcoin_gpu_hash_quad<<<grid, block>>>(threads, startNounce, d_resultNonce[thr_id], target);

    hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
