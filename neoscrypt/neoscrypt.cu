#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/neoscrypt.h"
}

#include "cuda_helper.h"
#include "miner.h"


static uint32_t *d_hash1[MAX_GPUS];
static uint32_t *d_hash2[MAX_GPUS]; // 2 streams

static uint32_t *t_hash1[MAX_GPUS];
static uint32_t *t_hash2[MAX_GPUS]; // 2 streams
static uint32_t *test[MAX_GPUS]; // 2 streams

static uint32_t *b_hash[MAX_GPUS];

extern void neoscrypt_setBlockTarget(uint32_t * data, const void *ptarget);
extern void neoscrypt_cpu_init(int thr_id, int threads, uint32_t* hash);
extern void neoscrypt_cpu_init_2stream(int thr_id, int threads, uint32_t* d_hash1, uint32_t* d_hash2, uint32_t* t_hash1, uint32_t* t_hash2, uint32_t* t_hash3, uint32_t* b_hash);


extern uint32_t neoscrypt_cpu_hash_k4(int stratum, int thr_id, int threads, uint32_t startNounce, int order);
extern uint32_t neoscrypt_cpu_hash_k4_2stream(int stratum, int thr_id, int threads, uint32_t startNounce, int order);

extern "C" int scanhash_neoscrypt(int stratum, int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x01ff;

	//	const int throughput = gpus_intensity[thr_id] ? 256 * 64 * gpus_intensity[thr_id] : 256 * 64 * 3.5;
	int intensity = (256 * 64 * 2);

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	if (strstr(props.name, "970"))
	{
		intensity = (256 * 64 * 5);
	}
	else if (strstr(props.name, "980"))
	{
#if CUDART_VERSION >= 7000
		intensity = (256 * 64 * 5);
#else
		intensity = (256 * 64 * 4);
#endif
	}
	else if (strstr(props.name, "750 Ti"))
	{
#if CUDART_VERSION >= 7000
		intensity = (256 * 64 * 3.5);
#else
		intensity = (256 * 64 * 3);
#endif
	}
	else if (strstr(props.name, "750"))
	{
		intensity = ((256 * 64 * 3.5) / 2);
	}
	else if (strstr(props.name, "960"))
	{
		intensity = (256 * 64 * 3.5);
	}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity) / 2;

	// throughput = min(throughput, (max_nonce - first_nonce));


	static bool init[MAX_GPUS] = { 0 };
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		//		hipDeviceReset();
		//		hipSetDeviceFlags(hipStreamNonBlocking);
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
#if CUDART_VERSION >= 7000
		CUDA_SAFE_CALL(hipMalloc(&d_hash1[thr_id], 32 * 130 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 32 * 130 * sizeof(uint64_t) * throughput));
#else
		CUDA_SAFE_CALL(hipMalloc(&d_hash1[thr_id], 32 * 130 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 32 * 130 * sizeof(uint64_t) * throughput));
#endif

		CUDA_SAFE_CALL(hipMalloc(&t_hash1[thr_id], 32 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&t_hash2[thr_id], 32 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&test[thr_id], 32 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&b_hash[thr_id], 128 * sizeof(uint32_t) * throughput));



		neoscrypt_cpu_init_2stream(thr_id, throughput, d_hash1[thr_id], d_hash2[thr_id], t_hash1[thr_id], t_hash2[thr_id], test[thr_id], b_hash[thr_id]);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	if (stratum) {
		for (int k = 0; k < 20; k++)
			be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}
	else {
		for (int k = 0; k < 20; k++)
			endiandata[k] = pdata[k];
	}



	neoscrypt_setBlockTarget(endiandata, ptarget);

	do {
		int order = 0;
		uint32_t foundNonce = neoscrypt_cpu_hash_k4_2stream(stratum, thr_id, throughput, pdata[19], order++);
		//		foundNonce = 10 + pdata[19];
		if (foundNonce != 0xffffffff)
		{
			if (opt_benchmark)
				applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, foundNonce);

			uint32_t vhash64[8];

			if (stratum) {
				be32enc(&endiandata[19], foundNonce);
			}
			else {
				endiandata[19] = foundNonce;
			}
			neoscrypt((unsigned char*)endiandata, (unsigned char*)vhash64, 0x80000620);

			if (vhash64[7] <= ptarget[7]) { // && fulltest(vhash64, ptarget)) {
				pdata[19] = foundNonce;
				*hashes_done = foundNonce - first_nonce + 1;
				return 1;
			}
			else {
				*hashes_done = foundNonce - first_nonce + 1; // keeps hashrate calculation happy
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}

		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
