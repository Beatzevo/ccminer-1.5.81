#include "hip/hip_runtime.h"


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB52 256
#define TPB50 64

 
#define Nrow 4
#define Ncol 4
#define u64type uint2
#define vectype uint28
#define memshift 3
__device__ vectype  *DMatrix;

 
__device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d = eorswap32 (a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

__device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}


 

__device__ __forceinline__ void reduceDuplex50(vectype state[4], uint32_t thread)
{
	const uint32_t ps1 = (Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * (Ncol - 1) + memshift * Ncol + Nrow * Ncol * memshift * thread);
	uint28 tmp[3];

//#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
#if __CUDA_ARCH__ == 500

		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 - i*memshift;

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]);

		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = __ldg4(&(DMatrix + s1)[j]) ^ state[j];
#else
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 - i*memshift;
		tmp[0] = __ldg4(&(DMatrix + s1)[0]);
		tmp[1] = __ldg4(&(DMatrix + s1)[1]);
		tmp[2] = __ldg4(&(DMatrix + s1)[2]);
		state[0] ^= tmp[0];
		state[1] ^= tmp[1];
		state[2] ^= tmp[2];

		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = tmp[j] ^ state[j];
#endif

	}
}
__device__  void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{

	int i, j;
		vectype state2[3],state1[3];

		const uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		const uint32_t ps3 = (memshift * (Ncol-1) + memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);
	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 - i*memshift;

		#if __CUDA_ARCH__ == 500
		#pragma unroll
		for (j = 0; j < 3; j++)
		{
			state[j] = state[j] ^ (__ldg4(&(DMatrix + s1)[j]) + __ldg4(&(DMatrix + s2)[j]));
		}
		
		round_lyra_v35(state);
#pragma unroll
		for (j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

#pragma unroll
		for (j = 0; j < 3; j++) 
		{
			(DMatrix + s3)[j] =state[j]^state1[j];
		}
		#else

#pragma unroll
		for (j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);
#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);
#pragma unroll
		for (j = 0; j < 3; j++)
		{
			state[j] ^= state1[j] + state2[j];
		}

		round_lyra_v35(state);

#pragma unroll
		for (j = 0; j < 3; j++)
		{			
			(DMatrix + s3)[j] = state1[j]^ state[j];;
		}

		#endif

		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		   #pragma unroll
		   for (j = 0; j < 11; j++)
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];


		#pragma unroll
		for (j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
	}
}



__device__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{
	int i,j;
		vectype state2[3];
		const uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		const uint32_t ps3 = (memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);
	
	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 + i*memshift;

		#pragma unroll 
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

		#pragma unroll 
		for (j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]) + state2[j];

		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		#pragma unroll 
		for (j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];
#if __CUDA_ARCH__ == 500
		if (rowInOut != rowOut) 
		{
			#pragma unroll 
			for ( j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} 
		if (rowInOut == rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
			state2[j] ^= state[j];
		}
#else
		if (rowInOut != rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} else
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				state2[j] ^= state[j];
		}
#endif

		#pragma unroll 
		for (j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(128, 1)
#endif
void lyra2v2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);



	vectype state[4];

	if (thread < threads)
	{
		const uint28 blake2b_IV[2] =
		{
			0xf3bcc908, 0x6a09e667,
			0x84caa73b, 0xbb67ae85,
			0xfe94f82b, 0x3c6ef372,
			0x5f1d36f1, 0xa54ff53a,
			0xade682d1, 0x510e527f,
			0x2b3e6c1f, 0x9b05688c,
			0xfb41bd6b, 0x1f83d9ab,
			0x137e2179, 0x5be0cd19
		};

		state[2] = ((blake2b_IV)[0]);
		state[3] = ((blake2b_IV)[1]);

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);

		 state[1] = state[0];

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);
		 ((uint2*)state)[0].x ^= 0x20;
		 ((uint2*)state)[1].x ^= 0x20;
		 ((uint2*)state)[2].x ^= 0x20;
		 ((uint2*)state)[3].x ^= 0x01;
		 ((uint2*)state)[4].x ^= 0x04;
		 ((uint2*)state)[5].x ^= 0x04;
		 ((uint2*)state)[6].x ^= 0x80;
		 ((uint2*)state)[7].y ^= 0x01000000;

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);

		const uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread);

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
		for (int i = 0; i < Ncol; i++)
		{
			const uint32_t s1 = ps1 - memshift * i;
			DMatrix[s1] = state[0];
			DMatrix[s1+1] = state[1];
			DMatrix[s1+2] = state[2];
			round_lyra_v35(state);
		}

		reduceDuplex50(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);

		uint32_t rowa;
		int prev=3;

        for (int i = 0; i < 4; i++)
        {
	     rowa = ((uint2*)state)[0].x & 3;  
		 reduceDuplexRowtV2(prev, rowa, i, state, thread);
         prev=i;
        }


		const uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        	round_lyra_v35(state);
		
		outputHash[thread] = ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2];
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, uint32_t tpb)
{
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	lyra2v2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
}

  