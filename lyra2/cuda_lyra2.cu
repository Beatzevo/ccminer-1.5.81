#include "hip/hip_runtime.h"


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB 8
//



#if __CUDA_ARCH__ == 500
#define u64type uint2
#define vectype uint28
#define memshift 3
#else 
#define u64type uint2
#define vectype uint28
#define memshift 4   
#endif
__device__ vectype  *DMatrix;

 
static __device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d = eorswap32(a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

static __device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}



static __device__ __forceinline__ void reduceDuplex(vectype state[4], uint32_t thread)
{


	    vectype state1[3]; 
		uint32_t ps1 = (256 * thread);
		uint32_t ps2 = (memshift * 7 + memshift * 8 + 256 * thread);

#pragma unroll 4
	for (int i = 0; i < 8; i++)
	{
        uint32_t s1 = ps1 + i*memshift;
        uint32_t s2 = ps2 - i*memshift;  
		
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]); 
 
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state); 
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexV3(vectype state[4], uint32_t thread)
{


	vectype state1[3];
	uint32_t ps1 = (256 * thread);
//                     colomn             row
	uint32_t ps2 = (memshift * 7 * 8 + memshift * 1 + 64 * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8 * i *memshift;
		uint32_t s2 = ps2 - 8 * i *memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];


		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


		vectype state2[3],state1[3];

		uint32_t ps1 = (              memshift * 8 * rowIn    + 256 * thread);
		uint32_t ps2 = (              memshift * 8 * rowInOut + 256 * thread);
		uint32_t ps3 = (memshift*7  + memshift * 8 * rowOut   + 256 * thread);


#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 - i*memshift;

		for (int j = 0; j < 3; j++) 
			state1[j]= __ldg4(&(DMatrix + s1)[j]);
		for (int j = 0; j < 3; j++)
			state2[j]= __ldg4(&(DMatrix + s2)[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}
		

		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}
 
		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++) 
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
		
	}


}

static __device__ __forceinline__ void reduceDuplexRowSetupV3(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


	vectype state2[3], state1[3];
	
	uint32_t ps1 = (                  memshift *  rowIn    + 64 * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut +                    64 * memshift* thread);
	uint32_t ps3 = (8 * memshift * 7 + memshift *  rowOut +  64 * memshift * thread);
	/*
	uint32_t ps1 = (256 * thread);
	uint32_t ps2 = (256 * thread);
	uint32_t ps3 = (256 * thread);
    */
#pragma nounroll 
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8*i*memshift;
		uint32_t s2 = ps2 + 8*i*memshift;
		uint32_t s3 = ps3 - 8*i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1 )[j]);
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2 )[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}


		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];

	}


}


static __device__ __forceinline__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

		vectype state1[3],state2[3];
		uint32_t ps1 = (memshift * 8 * rowIn + 256 * thread);
		uint32_t ps2 = (memshift * 8 * rowInOut + 256 * thread);
		uint32_t ps3 = (memshift * 8 * rowOut + 256 * thread);

#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 + i*memshift;


		for (int j = 0; j < 3; j++)  
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			          state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			          state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

if (rowInOut != rowOut) {

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j] = state2[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s3)[j] ^= state[j];

} else {

	for (int j = 0; j < 3; j++)
		state2[j] ^= state[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j]=state2[j];
}






	}
}

static __device__ __forceinline__ void reduceDuplexRowtV3(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

	vectype state1[3], state2[3];
	uint32_t ps1 = (memshift * rowIn + 64 * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut + 64 * memshift * thread);
	uint32_t ps3 = (memshift * rowOut + 64 *memshift * thread);

#pragma nounroll 
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8 * i*memshift;
		uint32_t s2 = ps2 + 8 * i*memshift;
		uint32_t s3 = ps3 + 8 * i*memshift;


		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

		if (rowInOut != rowOut) {

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		}
		else {

			for (int j = 0; j < 3; j++)
				state2[j] ^= state[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];
		}






	}
}



#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(48, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(16, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	   vectype state[4];
#if __CUDA_ARCH__ > 350
	const uint28 blake2b_IV[2] = {
		{{ 0xf3bcc908, 0x6a09e667 },
		{ 0x84caa73b, 0xbb67ae85 },
		{ 0xfe94f82b, 0x3c6ef372 },
		{ 0x5f1d36f1, 0xa54ff53a }},
		{{ 0xade682d1, 0x510e527f },
		{ 0x2b3e6c1f, 0x9b05688c },
		{ 0xfb41bd6b, 0x1f83d9ab },
		{ 0x137e2179, 0x5be0cd19 }}};
#else 
		const ulonglong4 blake2b_IV[2] = {
			{ 0x6a09e667f3bcc908,  
			  0xbb67ae8584caa73b,  
			  0x3c6ef372fe94f82b,  
			  0xa54ff53a5f1d36f1   },
			{ 0x510e527fade682d1,  
			  0x9b05688c2b3e6c1f,  
			  0x1f83d9abfb41bd6b,  
			  0x5be0cd19137e2179  } };
#endif
	
#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{
 
		 ((uint2*)state)[0] = __ldg(&outputHash[thread]);
		 ((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		 ((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		 ((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);
//		 state[0] = __ldg4(&((vectype*)outputHash)[thread]);
		 state[1] = state[0];
		 state[2] = ((vectype*)blake2b_IV)[0];
		 state[3] = ((vectype*)blake2b_IV)[1];

 
		for (int i = 0; i<24; i++) { round_lyra_v35(state); } //because 12 is not enough

             uint32_t ps1 = (memshift * 7  + 256 * thread);

		for (int i = 0; i < 8; i++)
		{
			uint32_t s1 = ps1 - memshift * i;
			for (int j = 0; j < 3; j++)
			    (DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplex(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);
		reduceDuplexRowSetupV2(3, 0, 4, state,  thread);
		reduceDuplexRowSetupV2(4, 3, 5, state,  thread);
		reduceDuplexRowSetupV2(5, 2, 6, state,  thread);
		reduceDuplexRowSetupV2(6, 1, 7, state,  thread);
		uint32_t rowa = ((uint2*)state)[0].x & 7;

		reduceDuplexRowtV2(7, rowa, 0, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(0, rowa, 3, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(3, rowa, 6, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(6, rowa, 1, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(1, rowa, 4, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(4, rowa, 7, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(7, rowa, 2, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(2, rowa, 5, state, thread);

		uint32_t shift = (memshift * 8 * rowa + 256 * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        			round_lyra_v35(state);
		

		outputHash[thread]=            ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2]; 
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}

#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(48, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(16, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2_gpu_hash_32_v3(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	vectype state[4];

#if __CUDA_ARCH__ > 350
	const uint28 blake2b_IV[2] = {
		{ { 0xf3bcc908, 0x6a09e667 },
		{ 0x84caa73b, 0xbb67ae85 },
		{ 0xfe94f82b, 0x3c6ef372 },
		{ 0x5f1d36f1, 0xa54ff53a } },
		{ { 0xade682d1, 0x510e527f },
		{ 0x2b3e6c1f, 0x9b05688c },
		{ 0xfb41bd6b, 0x1f83d9ab },
		{ 0x137e2179, 0x5be0cd19 } } };
#else 
	const ulonglong4 blake2b_IV[2] = {
		{ 0x6a09e667f3bcc908,
		0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b,
		0xa54ff53a5f1d36f1 },
		{ 0x510e527fade682d1,
		0x9b05688c2b3e6c1f,
		0x1f83d9abfb41bd6b,
		0x5be0cd19137e2179 } };
#endif


#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);
		
		state[1] = state[0];

		state[2] = ((vectype*)blake2b_IV)[0];
		state[3] = ((vectype*)blake2b_IV)[1];

		for (int i = 0; i<24; i++) 
                round_lyra_v35(state);  //because 12 is not enough

		uint32_t ps1 = (8 * memshift * 7 + 64 * memshift * thread);


		for (int i = 0; i < 8; i++)
		{
			uint32_t s1 = ps1 - 8 * memshift * i;
			for (int j = 0; j < 3; j++)
				(DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplexV3(state, thread);

		reduceDuplexRowSetupV3(1, 0, 2, state, thread);
		reduceDuplexRowSetupV3(2, 1, 3, state, thread);
		reduceDuplexRowSetupV3(3, 0, 4, state, thread);
		reduceDuplexRowSetupV3(4, 3, 5, state, thread);
		reduceDuplexRowSetupV3(5, 2, 6, state, thread);
		reduceDuplexRowSetupV3(6, 1, 7, state, thread);
		uint32_t rowa = ((uint2*)state)[0].x & 7;

		reduceDuplexRowtV3(7, rowa, 0, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(0, rowa, 3, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(3, rowa, 6, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(6, rowa, 1, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(1, rowa, 4, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(4, rowa, 7, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(7, rowa, 2, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(2, rowa, 5, state, thread);

		uint32_t shift = (memshift * rowa + 64 * memshift * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
			round_lyra_v35(state);


		outputHash[thread] = ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2];
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
		
	} //thread
}




__host__
void lyra2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
uint32_t tpb;
	if (device_sm[device_map[thr_id]]==500)
      tpb = 16; 
    else 
      tpb = TPB;
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	if (device_sm[device_map[thr_id]] == 500)
		lyra2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
    else 
    	lyra2_gpu_hash_32_v3 <<<grid, block>>> (threads, startNounce,(uint2*) d_outputHash);


}

  