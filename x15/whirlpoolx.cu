#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

#include "cuda_helper.h"

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern void cpu_whirlpoolx(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *foundNonce);
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{

	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	memset(hash, 0, sizeof(hash));

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

    
	for (uint32_t i = 0; i < 32; i++){
	        hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_whirlpoolx(int thr_id, uint32_t *pdata, uint32_t *ptarget, uint32_t max_nonce, uint32_t *hashes_done)
{
	uint32_t foundNonce[MAX_GPUS][4];
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, (1 << 25));
	throughput = min(throughput, max_nonce - first_nonce);
	if (opt_benchmark)
		ptarget[7] = 0x5;

	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		whirlpoolx_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
	{
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, &ptarget[6]);
	whirlpoolx_precompute(thr_id);
	do {
		cpu_whirlpoolx(thr_id, throughput, pdata[19], foundNonce[thr_id]);
//		CUDA_SAFE_CALL(hipGetLastError());
		if (foundNonce[thr_id][0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			/* check now with the CPU to confirm */
			be32enc(&endiandata[19], foundNonce[thr_id][0]);
			whirlxHash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
			/*		if (foundNonce[thr_id][1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], foundNonce[thr_id][1]);
					whirlxHash(vhash64, endiandata);
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[thr_id][1];
						res++;
						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nonce %08x", thr_id, foundNonce[thr_id][1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[thr_id][1]);
					}
				}
				*/

				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce %08x", thr_id, foundNonce[thr_id][0], vhash64[7]);
				pdata[19] = foundNonce[thr_id][0];
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[thr_id][0]);
			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
