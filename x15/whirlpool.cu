#include "hip/hip_runtime.h"
/*
 * whirlpool routine djm&SP
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}


#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);

extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
//extern uint32_t* whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);
extern void whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *foundnonce);


// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	// shavite 1
	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whc(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 20); // 19=256*256*8;
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id]) {
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		uint32_t foundNonce[2] = { 0, 0 };;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);

		whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], d_hash[thr_id], &foundNonce[0]);
		if (foundNonce[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			wcoinhash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], foundNonce[1]);
					wcoinhash(vhash64, endiandata);
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found second nounce %08x", thr_id, foundNonce[1]);
						pdata[21] = foundNonce[1];
						res++;
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[1]);
					}
				}
				
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nounce %08x", thr_id, foundNonce[0]);

				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[0]);
			}
		}
		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
