#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"



static uint32_t *d_gnounce[MAX_GPUS];
static uint32_t *d_GNonce[MAX_GPUS];

#define shl(x, n)            ((x) << (n))
#define shr(x, n)            ((x) >> (n))
//#define SHR(x, n) SHR2(x, n) 
//#define SHL(x, n) SHL2(x, n) 

#undef SPH_ROTL32
#define SPH_ROTL32 ROTL32


#define ROTL32host(x, n) (((x) << (n)) | ((x) >> (32 - (n))))
// #define SPH_ROTL32 SPH_ROTL32
#define ss0(x)  (shr((x), 1) ^ shl((x), 3) ^ SPH_ROTL32((x),  4) ^ SPH_ROTL32((x), 19))
#define ss1(x)  (shr((x), 1) ^ shl((x), 2) ^ __byte_perm(x,0,0x2103) ^ SPH_ROTL32((x), 23))
#define ss2(x)  (shr((x), 2) ^ shl((x), 1) ^ SPH_ROTL32((x), 12) ^ SPH_ROTL32((x), 25))
#define ss3(x)  (shr((x), 2) ^ shl((x), 2) ^ SPH_ROTL32((x), 15) ^ SPH_ROTL32((x), 29))
#define ss4(x)  (shr((x), 1) ^ (x))
#define ss5(x)  (shr((x), 2) ^ (x))
#define rs1(x) SPH_ROTL32((x),  3)
#define rs2(x) SPH_ROTL32((x),  7)
#define rs3(x) SPH_ROTL32((x), 13)
#define rs4(x) __byte_perm(x,0,0x1032)
#define rs5(x) SPH_ROTL32((x), 19)
#define rs6(x) SPH_ROTL32((x), 23)
#define rs7(x) SPH_ROTL32((x), 27)


/* Message expansion function 1 */
__forceinline__ __device__ uint32_t expand32_1(int i, uint32_t *M32,const uint32_t *H, uint32_t *Q)
{
	return (ss1(Q[i - 16]) + ss2(Q[i - 15]) + ss3(Q[i - 14]) + ss0(Q[i - 13])
		+ ss1(Q[i - 12]) + ss2(Q[i - 11]) + ss3(Q[i - 10]) + ss0(Q[i - 9])
		+ ss1(Q[i - 8]) + ss2(Q[i - 7]) + ss3(Q[i - 6]) + ss0(Q[i - 5])
		+ ss1(Q[i - 4]) + ss2(Q[i - 3]) + ss3(Q[i - 2]) + ss0(Q[i - 1])
		+ ((i*(0x05555555ul) + SPH_ROTL32(M32[(i - 16) % 16], ((i - 16) % 16) + 1) + SPH_ROTL32(M32[(i - 13) % 16], ((i - 13) % 16) + 1) - SPH_ROTL32(M32[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ H[(i - 16 + 7) % 16]));
}

/* Message expansion function 2 */
__forceinline__ __device__ uint32_t expand32_2(const int i, uint32_t *M32, const uint32_t *H, uint32_t *Q)
{
	return (
		rs2(Q[i - 13]) + rs3(Q[i - 11]) + rs4(Q[i - 9]) + rs1(Q[i - 15])+
		+rs5(Q[i - 7]) + rs6(Q[i - 5]) + rs7(Q[i - 3]) + ss4(Q[i - 2]) + ss5(Q[i - 1]));
}

__forceinline__ __device__ void Compression256(uint32_t *  M32)
{
	const uint32_t H[16] = {
		(0x40414243), (0x44454647),
		(0x48494A4B), (0x4C4D4E4F),
		(0x50515253), (0x54555657),
		(0x58595A5B), (0x5C5D5E5F),
		(0x60616263), (0x64656667),
		(0x68696A6B), (0x6C6D6E6F),
		(0x70717273), (0x74757677),
		(0x78797A7B), (0x7C7D7E7F)
	};

	M32[8] = 0x80;
	M32[14] = 0x100;

//	int i;
	uint32_t XL32, XH32, Q[32];

	Q[0] = (M32[5] ^ H[5]) - (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[13] ^ H[13]) + (M32[14] ^ H[14]);
	Q[1] = (M32[6] ^ H[6]) - (M32[8] ^ H[8]) + (M32[11] ^ H[11]) + (M32[14] ^ H[14]) - (M32[15] ^ H[15]);
	Q[2] = (M32[0] ^ H[0]) + (M32[7] ^ H[7]) + (M32[9] ^ H[9]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[3] = (M32[0] ^ H[0]) - (M32[1] ^ H[1]) + (M32[8] ^ H[8]) - (M32[10] ^ H[10]) + (M32[13] ^ H[13]);
	Q[4] = (M32[1] ^ H[1]) + (M32[2] ^ H[2]) + (M32[9] ^ H[9]) - (M32[11] ^ H[11]) - (M32[14] ^ H[14]);
	Q[5] = (M32[3] ^ H[3]) - (M32[2] ^ H[2]) + (M32[10] ^ H[10]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[6] = (M32[4] ^ H[4]) - (M32[0] ^ H[0]) - (M32[3] ^ H[3]) - (M32[11] ^ H[11]) + (M32[13] ^ H[13]);
	Q[7] = (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[5] ^ H[5]) - (M32[12] ^ H[12]) - (M32[14] ^ H[14]);
	Q[8] = (M32[2] ^ H[2]) - (M32[5] ^ H[5]) - (M32[6] ^ H[6]) + (M32[13] ^ H[13]) - (M32[15] ^ H[15]);
	Q[9] = (M32[0] ^ H[0]) - (M32[3] ^ H[3]) + (M32[6] ^ H[6]) - (M32[7] ^ H[7]) + (M32[14] ^ H[14]);
	Q[10] = (M32[8] ^ H[8]) - (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[7] ^ H[7]) + (M32[15] ^ H[15]);
	Q[11] = (M32[8] ^ H[8]) - (M32[0] ^ H[0]) - (M32[2] ^ H[2]) - (M32[5] ^ H[5]) + (M32[9] ^ H[9]);
	Q[12] = (M32[1] ^ H[1]) + (M32[3] ^ H[3]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[10] ^ H[10]);
	Q[13] = (M32[2] ^ H[2]) + (M32[4] ^ H[4]) + (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[11] ^ H[11]);
	Q[14] = (M32[3] ^ H[3]) - (M32[5] ^ H[5]) + (M32[8] ^ H[8]) - (M32[11] ^ H[11]) - (M32[12] ^ H[12]);
	Q[15] = (M32[12] ^ H[12]) - (M32[4] ^ H[4]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[13] ^ H[13]);

	/*  Diffuse the differences in every word in a bijective manner with ssi, and then add the values of the previous double pipe.*/
	Q[0] = ss0(Q[0]) + H[1];
	Q[1] = ss1(Q[1]) + H[2];
	Q[2] = ss2(Q[2]) + H[3];
	Q[3] = ss3(Q[3]) + H[4];
	Q[4] = ss4(Q[4]) + H[5];
	Q[5] = ss0(Q[5]) + H[6];
	Q[6] = ss1(Q[6]) + H[7];
	Q[7] = ss2(Q[7]) + H[8];
	Q[8] = ss3(Q[8]) + H[9];
	Q[9] = ss4(Q[9]) + H[10];
	Q[10] = ss0(Q[10]) + H[11];
	Q[11] = ss1(Q[11]) + H[12];
	Q[12] = ss2(Q[12]) + H[13];
	Q[13] = ss3(Q[13]) + H[14];
	Q[14] = ss4(Q[14]) + H[15];
	Q[15] = ss0(Q[15]) + H[0];

	/* This is the Message expansion or f_1 in the documentation.       */
	/* It has 16 rounds.                                                */
	/* Blue Midnight Wish has two tunable security parameters.          */
	/* The parameters are named EXPAND_1_ROUNDS and EXPAND_2_ROUNDS.    */
	/* The following relation for these parameters should is satisfied: */
	/* EXPAND_1_ROUNDS + EXPAND_2_ROUNDS = 16                           */

//	#pragma unroll
//	for (i = 0; i<2; i++)
//		Q[i + 16] = expand32_1(i + 16, M32, H, Q);

	Q[16]=ss1(Q[16 - 16]) + ss2(Q[16 - 15]) + ss3(Q[16 - 14]) + ss0(Q[16 - 13])
		+ ss1(Q[16 - 12]) + ss2(Q[16 - 11]) + ss3(Q[16 - 10]) + ss0(Q[16 - 9])
		+ ss1(Q[16 - 8]) + ss2(Q[16 - 7]) + ss3(Q[16 - 6]) + ss0(Q[16 - 5])
		+ ss1(Q[16 - 4]) + ss2(Q[16 - 3]) + ss3(Q[16 - 2]) + ss0(Q[16 - 1])
		+ ((16*(0x05555555ul) + SPH_ROTL32(M32[0], ((16 - 16) % 16) + 1) + SPH_ROTL32(M32[3], ((16 - 13) % 16) + 1)) ^ H[(16 - 16 + 7) % 16]);

	Q[17] = ss1(Q[17 - 16]) + ss2(Q[17 - 15]) + ss3(Q[17 - 14]) + ss0(Q[17 - 13])
	 		+ ss1(Q[17 - 12]) + ss2(Q[17 - 11]) + ss3(Q[17 - 10]) + ss0(Q[17 - 9])
			+ ss1(Q[17 - 8]) + ss2(Q[17 - 7]) + ss3(Q[17 - 6]) + ss0(Q[17 - 5])
			+ ss1(Q[17 - 4]) + ss2(Q[17 - 3]) + ss3(Q[17 - 2]) + ss0(Q[17 - 1])
			+ ((17 * (0x05555555ul) + SPH_ROTL32(M32[(17 - 16) % 16], ((17 - 16) % 16) + 1) + SPH_ROTL32(M32[(17 - 13) % 16], ((17 - 13) % 16) + 1)) ^ H[(17 - 16 + 7) % 16]);


	uint32_t precalc = Q[18 - 16] + Q[18 - 14] + Q[18 - 12] + Q[18 - 10] + Q[18 - 8] + Q[18 - 6] ; //+ Q[18 - 4]
	uint32_t precalc2 = Q[19 - 16] + Q[19 - 14] + Q[19 - 12] + Q[19 - 10] + Q[19 - 8] + Q[19 - 6] ;//+ Q[19 - 4]

//	#pragma unroll
//	for (i = 2 + 16; i < 16 + 16; i+=2)
//	{
		precalc = precalc + Q[18 - 4];
		precalc2 = precalc2 + Q[18 + 1 - 4];
		uint32_t p1 = ((18 * (0x05555555ul) + SPH_ROTL32(M32[2], ((18 - 16) % 16) + 1) + SPH_ROTL32(M32[5], ((18 - 13) % 16) + 1)) ^ H[(18 - 16 + 7) % 16]);
		uint32_t p2 = (((18 + 1)*(0x05555555ul) + SPH_ROTL32(M32[3], (((18 + 1) - 16) % 16) + 1) + SPH_ROTL32(M32[6], (((18 + 1) - 13) % 16) + 1)) ^ H[((18 + 1) - 16 + 7) % 16]);
		Q[18] = precalc + expand32_2(18, M32, H, Q) + p1;
		Q[18 + 1] = precalc2 + expand32_2(18 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[18 - 16];
		precalc2 = precalc2 - Q[18 + 1 - 16];

		precalc = precalc + Q[20 - 4];
		precalc2 = precalc2 + Q[20 + 1 - 4];
		p1 = ((20 * (0x05555555ul) + SPH_ROTL32(M32[4], ((20 - 16) % 16) + 1) + SPH_ROTL32(M32[7], ((20 - 13) % 16) + 1) - (0x100<< 15)) ^ H[(20 - 16 + 7) % 16]);
		p2 = (((20 + 1)*(0x05555555ul) + SPH_ROTL32(M32[5], (((20 + 1) - 16) % 16) + 1) + (0x80<<9)) ^ H[((20 + 1) - 16 + 7) % 16]);
		Q[20] = precalc + expand32_2(20, M32, H, Q) + p1;
		Q[20 + 1] = precalc2 + expand32_2(20 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[20 - 16];
		precalc2 = precalc2 - Q[20 + 1 - 16];

		precalc = precalc + Q[22 - 4];
		precalc2 = precalc2 + Q[22 + 1 - 4];
		p1 = ((22 * (0x05555555ul) + SPH_ROTL32(M32[6], ((22 - 16) % 16) + 1) - SPH_ROTL32(M32[0], ((22 - 6) % 16) + 1)) ^ H[(22 - 16 + 7) % 16]);
		p2 = (((22 + 1)*(0x05555555ul) + SPH_ROTL32(M32[7], (((22 + 1) - 16) % 16) + 1) - SPH_ROTL32(M32[1], (((22 + 1) - 6) % 16) + 1)) ^ H[((22 + 1) - 16 + 7) % 16]);
		Q[22] = precalc + expand32_2(22, M32, H, Q) + p1;
		Q[22 + 1] = precalc2 + expand32_2(22 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[22 - 16];
		precalc2 = precalc2 - Q[22 + 1 - 16];

		precalc = precalc + Q[24 - 4];
		precalc2 = precalc2 + Q[24 + 1 - 4];
		p1 = ((24 * (0x05555555ul) + (0x80 << 9) - SPH_ROTL32(M32[2], ((24 - 6) % 16) + 1)) ^ H[(24 - 16 + 7) % 16]);
		p2 = (((24 + 1)*(0x05555555ul) - SPH_ROTL32(M32[3], (((24 + 1) - 6) % 16) + 1)) ^ H[((24 + 1) - 16 + 7) % 16]);
		Q[24] = precalc + expand32_2(24, M32, H, Q) + p1;
		Q[24 + 1] = precalc2 + expand32_2(24 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[24 - 16];
		precalc2 = precalc2 - Q[24 + 1 - 16];

		precalc = precalc + Q[26 - 4];
		precalc2 = precalc2 + Q[26 + 1 - 4];
		p1 = ((26 * (0x05555555ul) - SPH_ROTL32(M32[4], ((26 - 6) % 16) + 1)) ^ H[(26 - 16 + 7) % 16]);
		p2 = (((26 + 1)*(0x05555555ul) + (0x100 << 15) - SPH_ROTL32(M32[5], (((26 + 1) - 6) % 16) + 1)) ^ H[((26 + 1) - 16 + 7) % 16]);
		Q[26] = precalc + expand32_2(26, M32, H, Q) + p1;
		Q[26 + 1] = precalc2 + expand32_2(26 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[26 - 16];
		precalc2 = precalc2 - Q[26 + 1 - 16];

		precalc = precalc + Q[28 - 4];
		precalc2 = precalc2 + Q[28 + 1 - 4];
		p1 = ((28 * (0x05555555ul) - SPH_ROTL32(M32[6], ((28 - 6) % 16) + 1)) ^ H[(28 - 16 + 7) % 16]);
		p2 = (((28 + 1)*(0x05555555ul) + SPH_ROTL32(M32[0], (((28 + 1) - 13) % 16) + 1) - SPH_ROTL32(M32[7], (((28 + 1) - 6) % 16) + 1)) ^ H[((28 + 1) - 16 + 7) % 16]);
		Q[28] = precalc + expand32_2(28, M32, H, Q) + p1;
		Q[28 + 1] = precalc2 + expand32_2(28 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[28 - 16];
		precalc2 = precalc2 - Q[28 + 1 - 16];

		precalc = precalc + Q[30 - 4];
		precalc2 = precalc2 + Q[30 + 1 - 4];
		p1 = ((30 * (0x05555555ul) + (0x100 << 15) + SPH_ROTL32(M32[1], ((30 - 13) % 16) + 1) - (0x80 << 9)) ^ H[(30 - 16 + 7) % 16]);
		p2 = (((30 + 1)*(0x05555555ul) + SPH_ROTL32(M32[2], (((30 + 1) - 13) % 16) + 1)) ^ H[((30 + 1) - 16 + 7) % 16]);
		Q[30] = precalc + expand32_2(30, M32, H, Q) + p1;
		Q[30 + 1] = precalc2 + expand32_2(30 + 1, M32, H, Q) + p2;
		precalc = precalc - Q[30 - 16];
		precalc2 = precalc2 - Q[30 + 1 - 16];

	/* Blue Midnight Wish has two temporary cummulative variables that accumulate via XORing */
	/* 16 new variables that are prooduced in the Message Expansion part.                    */
	XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
	XH32 = XL32^Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];


	/*  This part is the function f_2 - in the documentation            */

	/*  Compute the double chaining pipe for the next message block.    */
	M32[0] = (shl(XH32, 5) ^ shr(Q[16], 5) ^ M32[0]) + (XL32    ^ Q[24] ^ Q[0]);
	M32[1] = (shr(XH32, 7) ^ shl(Q[17], 8) ^ M32[1]) + (XL32    ^ Q[25] ^ Q[1]);
	M32[2] = (shr(XH32, 5) ^ shl(Q[18], 5) ^ M32[2]) + (XL32    ^ Q[26] ^ Q[2]);
	M32[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ M32[3]) + (XL32    ^ Q[27] ^ Q[3]);
	M32[4] = (shr(XH32, 3) ^ Q[20] ^ M32[4]) + (XL32    ^ Q[28] ^ Q[4]);
	M32[5] = (shl(XH32, 6) ^ shr(Q[21], 6) ^ M32[5]) + (XL32    ^ Q[29] ^ Q[5]);
	M32[6] = (shr(XH32, 4) ^ shl(Q[22], 6) ^ M32[6]) + (XL32    ^ Q[30] ^ Q[6]);
	M32[7] = (shr(XH32, 11) ^ shl(Q[23], 2) ^ M32[7]) + (XL32    ^ Q[31] ^ Q[7]);

	M32[8] = SPH_ROTL32(M32[4], 9) + (XH32     ^     Q[24] ^ M32[8]) + (shl(XL32, 8) ^ Q[23] ^ Q[8]);
	M32[9] = SPH_ROTL32(M32[5], 10) + (XH32     ^     Q[25] ^ M32[9]) + (shr(XL32, 6) ^ Q[16] ^ Q[9]);
	M32[10] = SPH_ROTL32(M32[6], 11) + (XH32     ^     Q[26] ^ M32[10]) + (shl(XL32, 6) ^ Q[17] ^ Q[10]);
	M32[11] = SPH_ROTL32(M32[7], 12) + (XH32     ^     Q[27] ^ M32[11]) + (shl(XL32, 4) ^ Q[18] ^ Q[11]);
	M32[12] = SPH_ROTL32(M32[0], 13) + (XH32     ^     Q[28] ^ M32[12]) + (shr(XL32, 3) ^ Q[19] ^ Q[12]);
	M32[13] = SPH_ROTL32(M32[1], 14) + (XH32     ^     Q[29] ^ M32[13]) + (shr(XL32, 4) ^ Q[20] ^ Q[13]);
	M32[14] = SPH_ROTL32(M32[2], 15) + (XH32     ^     Q[30] ^ M32[14]) + (shr(XL32, 7) ^ Q[21] ^ Q[14]);
	M32[15] = SPH_ROTL32(M32[3], 16) + (XH32     ^     Q[31] ^ M32[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);
}

__forceinline__ __device__ void Compression256_2(uint32_t *  M32)
{
	const uint32_t H[16] = {
		(0xaaaaaaa0), (0xaaaaaaa1), (0xaaaaaaa2),
		(0xaaaaaaa3), (0xaaaaaaa4), (0xaaaaaaa5),
		(0xaaaaaaa6), (0xaaaaaaa7), (0xaaaaaaa8),
		(0xaaaaaaa9), (0xaaaaaaaa), (0xaaaaaaab),
		(0xaaaaaaac), (0xaaaaaaad), (0xaaaaaaae),
		(0xaaaaaaaf)
	};
	int i;
	uint32_t XL32, XH32, Q[32];

	Q[0] = (M32[5] ^ H[5]) - (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[13] ^ H[13]) + (M32[14] ^ H[14]);
	Q[1] = (M32[6] ^ H[6]) - (M32[8] ^ H[8]) + (M32[11] ^ H[11]) + (M32[14] ^ H[14]) - (M32[15] ^ H[15]);
	Q[2] = (M32[0] ^ H[0]) + (M32[7] ^ H[7]) + (M32[9] ^ H[9]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[3] = (M32[0] ^ H[0]) - (M32[1] ^ H[1]) + (M32[8] ^ H[8]) - (M32[10] ^ H[10]) + (M32[13] ^ H[13]);
	Q[4] = (M32[1] ^ H[1]) + (M32[2] ^ H[2]) + (M32[9] ^ H[9]) - (M32[11] ^ H[11]) - (M32[14] ^ H[14]);
	Q[5] = (M32[3] ^ H[3]) - (M32[2] ^ H[2]) + (M32[10] ^ H[10]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[6] = (M32[4] ^ H[4]) - (M32[0] ^ H[0]) - (M32[3] ^ H[3]) - (M32[11] ^ H[11]) + (M32[13] ^ H[13]);
	Q[7] = (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[5] ^ H[5]) - (M32[12] ^ H[12]) - (M32[14] ^ H[14]);
	Q[8] = (M32[2] ^ H[2]) - (M32[5] ^ H[5]) - (M32[6] ^ H[6]) + (M32[13] ^ H[13]) - (M32[15] ^ H[15]);
	Q[9] = (M32[0] ^ H[0]) - (M32[3] ^ H[3]) + (M32[6] ^ H[6]) - (M32[7] ^ H[7]) + (M32[14] ^ H[14]);
	Q[10] = (M32[8] ^ H[8]) - (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[7] ^ H[7]) + (M32[15] ^ H[15]);
	Q[11] = (M32[8] ^ H[8]) - (M32[0] ^ H[0]) - (M32[2] ^ H[2]) - (M32[5] ^ H[5]) + (M32[9] ^ H[9]);
	Q[12] = (M32[1] ^ H[1]) + (M32[3] ^ H[3]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[10] ^ H[10]);
	Q[13] = (M32[2] ^ H[2]) + (M32[4] ^ H[4]) + (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[11] ^ H[11]);
	Q[14] = (M32[3] ^ H[3]) - (M32[5] ^ H[5]) + (M32[8] ^ H[8]) - (M32[11] ^ H[11]) - (M32[12] ^ H[12]);
	Q[15] = (M32[12] ^ H[12]) - (M32[4] ^ H[4]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[13] ^ H[13]);

	/*  Diffuse the differences in every word in a bijective manner with ssi, and then add the values of the previous double pipe.*/
	Q[0] = ss0(Q[0]) + H[1];
	Q[1] = ss1(Q[1]) + H[2];
	Q[2] = ss2(Q[2]) + H[3];
	Q[3] = ss3(Q[3]) + H[4];
	Q[4] = ss4(Q[4]) + H[5];
	Q[5] = ss0(Q[5]) + H[6];
	Q[6] = ss1(Q[6]) + H[7];
	Q[7] = ss2(Q[7]) + H[8];
	Q[8] = ss3(Q[8]) + H[9];
	Q[9] = ss4(Q[9]) + H[10];
	Q[10] = ss0(Q[10]) + H[11];
	Q[11] = ss1(Q[11]) + H[12];
	Q[12] = ss2(Q[12]) + H[13];
	Q[13] = ss3(Q[13]) + H[14];
	Q[14] = ss4(Q[14]) + H[15];
	Q[15] = ss0(Q[15]) + H[0];

	/* This is the Message expansion or f_1 in the documentation.       */
	/* It has 16 rounds.                                                */
	/* Blue Midnight Wish has two tunable security parameters.          */
	/* The parameters are named EXPAND_1_ROUNDS and EXPAND_2_ROUNDS.    */
	/* The following relation for these parameters should is satisfied: */
	/* EXPAND_1_ROUNDS + EXPAND_2_ROUNDS = 16                           */

	#pragma unroll
	for (i = 0; i<2; i++)
		Q[i + 16] = expand32_1(i + 16, M32, H, Q);

/*	#pragma unroll
	for (i = 2; i<16; i++)
		Q[i + 16] = expand32_2(i + 16, M32, H, Q);
*/
	uint32_t precalc = Q[18 - 16] + Q[18 - 14] + Q[18 - 12] + Q[18 - 10] + Q[18 - 8] + Q[18 - 6] ; //+ Q[18 - 4]
	uint32_t precalc2 = Q[19 - 16] + Q[19 - 14] + Q[19 - 12] + Q[19 - 10] + Q[19 - 8] + Q[19 - 6] ;//+ Q[19 - 4]

	#pragma unroll
	for (i = 2 + 16; i < 16 + 16; i+=2)
	{
		precalc = precalc + Q[i - 4];
		precalc2 = precalc2 + Q[i + 1 - 4];
		uint32_t p1 = ((i*(0x05555555ul) + SPH_ROTL32(M32[(i - 16) % 16], ((i - 16) % 16) + 1) + SPH_ROTL32(M32[(i - 13) % 16], ((i - 13) % 16) + 1) - SPH_ROTL32(M32[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ H[(i - 16 + 7) % 16]);
		uint32_t p2 = (((i + 1)*(0x05555555ul) + SPH_ROTL32(M32[((i + 1) - 16) % 16], (((i + 1) - 16) % 16) + 1) + SPH_ROTL32(M32[((i + 1) - 13) % 16], (((i + 1) - 13) % 16) + 1) - SPH_ROTL32(M32[((i + 1) - 6) % 16], (((i + 1) - 6) % 16) + 1)) ^ H[((i + 1) - 16 + 7) % 16]);
		Q[i] = precalc + expand32_2(i, M32, H, Q) + p1;
		Q[i + 1] = precalc2 + expand32_2(i + 1, M32, H, Q) + p2;
		precalc = precalc - Q[i - 16];
		precalc2 = precalc2 - Q[i + 1 - 16];
	}



	/* Blue Midnight Wish has two temporary cummulative variables that accumulate via XORing */
	/* 16 new variables that are prooduced in the Message Expansion part.                    */
	XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
	XH32 = XL32^Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];


	M32[2] = (shr(XH32, 5) ^ shl(Q[18], 5) ^ M32[2]) + (XL32    ^ Q[26] ^ Q[2]);
	M32[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ M32[3]) + (XL32    ^ Q[27] ^ Q[3]);
	M32[14] = SPH_ROTL32(M32[2], 15) + (XH32     ^     Q[30] ^ M32[14]) + (shr(XL32, 7) ^ Q[21] ^ Q[14]);
	M32[15] = SPH_ROTL32(M32[3], 16) + (XH32     ^     Q[31] ^ M32[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);


}

#define TPB 64
__global__ __launch_bounds__(TPB,16)
void bmw256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *const __restrict__ nonceVector, uint32_t Target)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t backup = Target;
		uint2 message[8]={0};
		uint32_t nonce = (startNounce + thread);


		message[0] = __ldg(&g_hash[thread]);
		message[1] = __ldg(&g_hash[thread + 1 * threads]);
		message[2] = __ldg(&g_hash[thread + 2 * threads]);
		message[3] = __ldg(&g_hash[thread + 3 * threads]);


		Compression256((uint32_t *)message);
		Compression256_2((uint32_t *)message);

		if (message[7].y <= backup)
		{
			uint32_t tmp = atomicCAS(nonceVector, 0xffffffff, nonce);
			if (tmp != 0xffffffff)
				nonceVector[1] = nonce;
		}
	}
}


__host__
void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t Target)
{
	hipMemset(d_GNonce[thr_id], 0xffffffff, 2 * sizeof(uint32_t));

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + TPB - 1) / TPB);
	dim3 block(TPB);

	bmw256_gpu_hash_32 << <grid, block >> >(threads, startNounce, (uint2 *)g_hash, d_GNonce[thr_id],Target);
	hipMemcpy(d_gnounce[thr_id], d_GNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	resultnonces[0] = *(d_gnounce[thr_id]);
	resultnonces[1] = *(d_gnounce[thr_id] + 1);
}


__host__
void bmw256_cpu_init(int thr_id, uint32_t threads)
{
	hipMalloc(&d_GNonce[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&d_gnounce[thr_id], 2 * sizeof(uint32_t));
}

/*
__host__
void bmw256_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}
*/