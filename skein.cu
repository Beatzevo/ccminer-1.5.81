#include "hip/hip_runtime.h"
/**
* SKEIN512 80 + SHA256 64
* by tpruvot@github - 2015
* Optimized by sp-hash@github - 2015
*/

extern "C" {
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include <openssl/sha.h>
 
static uint32_t foundnonces[MAX_GPUS][2];

extern void skein512_cpu_setBlock_80(uint32_t thr_id,void *pdata);
extern void skein512_cpu_hash_80_50(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint2 target, uint32_t *h_found);
extern void skein512_cpu_hash_80_52(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint2 target, uint32_t *h_found);

extern "C" void skeincoinhash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;
	SHA256_CTX sha256;

	uint32_t hash[16];

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, hash);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char *)hash, 64);
	SHA256_Final((unsigned char *)hash, &sha256);

	memcpy(output, hash, 32);
}

static __inline uint32_t swab32_if(uint32_t val, bool iftrue)
{
	return iftrue ? swab32(val) : val;
}

bool init[MAX_GPUS] = { 0 };

int scanhash_skeincoin(int thr_id, uint32_t *pdata,
								  const uint32_t *ptarget, uint32_t max_nonce,
								  uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	const bool swap = true;

	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 26 : 1 << 25;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
	{
		((uint64_t*)ptarget)[3] = 0x3000f0000;
	}
//	uint64_t target = ((uint64_t*)ptarget)[3];
	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(thr_id, (void*)endiandata);
	do
	{
//		if(scan_abort_flag || work_restart[thr_id].restart) return 0;
		if (device_sm[device_map[thr_id]] > 500)
			skein512_cpu_hash_80_52(thr_id, throughput, pdata[19], swap, ((uint2*)ptarget)[3], foundnonces[thr_id]);
		else
			skein512_cpu_hash_80_50(thr_id, throughput, pdata[19], swap, ((uint2*)ptarget)[3], foundnonces[thr_id]);
		
		if (foundnonces[thr_id][0] != 0xffffffff)
		{
			uint32_t vhash64[8];

			endiandata[19] = swab32_if(foundnonces[thr_id][0], swap);
			
			skeincoinhash(vhash64, endiandata);

			uint64_t test = ((uint64_t*)vhash64)[3];
			if (test <= ((uint64_t*)ptarget)[3] && fulltest(vhash64, ptarget))
			{
				int res = 1;

				if (opt_debug || opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce $%08X", thr_id, foundnonces[thr_id][0]);
				if (foundnonces[thr_id][1] != 0xffffffff)
				{
					if (foundnonces[thr_id][1] == foundnonces[thr_id][0])
					{
						//applog(LOG_WARNING, "Duplicate nonce: #%d", test);
					}
					else
					{
						if (opt_debug || opt_benchmark)
							applog(LOG_INFO, "GPU #%d: found second nonce $%08X", thr_id, foundnonces[thr_id][1]);
						pdata[21] = swab32_if(foundnonces[thr_id][1], !swap);
						res++;
					}
				}
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = swab32_if(foundnonces[thr_id][0], !swap);
				return res;
			}
			else 
			{
				if (test != ((uint64_t*)ptarget)[3])
					applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces[thr_id][0]);
				else
					applog(LOG_WARNING, "Lost work: #%d", test);

			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
